/**
 * @author  Viktor-Adam Koropecky
 */

#include "mex.h"
#include "stdio.h"
#include "stdint.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include <chrono>
#include "Kernels.h"

// Cutting the input image on CPU by copying partial columns 
void h_cut( int M, int N, int offsets_y, int offsets_x, int dimensions_y, int dimensions_x, uint8_t* input, uint8_t* resize ){
    int count = dimensions_x;
    for(int i = 0; i < count; i+=1){
        memcpy(&resize[i * dimensions_y], &input[offsets_y + M * offsets_x + M * i], sizeof(uint8_t)*dimensions_y);
        memcpy(&resize[i * dimensions_y + dimensions_y*dimensions_x], &input[offsets_y + M * offsets_x + M * i + M*N], sizeof(uint8_t)*dimensions_y);
        memcpy(&resize[i * dimensions_y + dimensions_y*dimensions_x*2], &input[offsets_y + M * offsets_x + M * i + 2*M*N], sizeof(uint8_t)*dimensions_y);
    }
}

void mexFunction(int nlhs, mxArray *plhs[],
    int nrhs, const mxArray *prhs[])
{
    std::chrono::steady_clock::time_point start = std::chrono::steady_clock::now();

    #define Erz plhs[0]

    #define raw_data prhs[0]
    #define lambda mxGetPr(prhs[1])[0]
    #define n mxGetPr(prhs[2])[0]
    #define dx mxGetPr(prhs[3])[0]
    #define z mxGetPr(prhs[4])[0]
    #define offsts prhs[5]
    #define dims prhs[6]

    int M = (int)mxGetDimensions(raw_data)[0];
    int N = (int)mxGetDimensions(raw_data)[1];
    int Z = (int)mxGetDimensions(raw_data)[2];

    int blockSize = 1024;
    int numBlocks;
    
    uint8_t* input;
    uint8_t* h_input;

    uint16_t* RGB;
    uint16_t* R;
    uint16_t* G;
    uint16_t* B;
    uint16_t* temporary;

    hipfftHandle plan;

    hipfftComplex* doubleComplexArray;
    hipfftComplex* Hq;
    hipfftComplex* image;

    float* doubleArray;
    float* outputArray;
    float* smallerOutputArray;
    float* doubleTemporary;

    /* 
        Input data setup. Program checks whether it is necessary to cut the input data and change parameters.
    */
    if(nrhs == 7){
        int* offsets = (int*)mxGetPr(offsts);
        int* dimensions = (int*)mxGetPr(dims);

        offsets[1] = ((offsets[1] % 2) == 1) ? offsets[1]-1 : offsets[1];
        dimensions[1] = ((dimensions[1] % 2) == 1) ? dimensions[1]-1 : dimensions[1];
        
        h_input = (uint8_t*)malloc(dimensions[1]*dimensions[0]*Z*sizeof(uint8_t));

        h_cut(M, N,offsets[0],offsets[1], dimensions[0],dimensions[1], (uint8_t*)mxGetData(raw_data), h_input);

        M = dimensions[0];
        N = dimensions[1];

        hipMalloc(&input, N*M*Z*sizeof(uint8_t));
        hipMemcpy(input, h_input, Z*N*M*sizeof(uint8_t), hipMemcpyHostToDevice);
    }
    else{
        hipMalloc(&input, N*M*Z*sizeof(uint8_t));
        hipMemcpy(input, (uint8_t*)mxGetData(raw_data), Z*N*M*sizeof(uint8_t), hipMemcpyHostToDevice);
    }
    std::chrono::steady_clock::time_point setup = std::chrono::steady_clock::now();

    /*
        Allocation of memory. We allocate larger arrays so we can call hipMalloc less.
    */
    hipMalloc(&RGB, 4*N*M*sizeof(uint16_t));
    R = &RGB[0];
    G = &RGB[N*M];
    B = &RGB[2*N*M];
    temporary = &RGB[3*N*M];

    hipMalloc(&doubleComplexArray, 2*N*M*sizeof(hipfftComplex));
    Hq = &doubleComplexArray[0];
    image = &doubleComplexArray[N*M];

    hipMalloc(&doubleArray, 2*N*M*sizeof(float));
    doubleTemporary = &doubleArray[0];
    outputArray = &doubleArray[N*M];

    hipMalloc(&smallerOutputArray, N*M*sizeof(float)/4);

    // Declaration of output matrix.
    Erz = mxCreateNumericMatrix(M,N,mxSINGLE_CLASS,mxREAL); 
    std::chrono::steady_clock::time_point mallocs = std::chrono::steady_clock::now();

    // Declaring appropriate number of cuda Blocks.
    numBlocks = (N*M/2 +blockSize -1)/blockSize;
    // Transforming camera data to a Bayer image.
    bayerize<<<numBlocks,blockSize>>>(M, N, input, temporary);
    std::chrono::steady_clock::time_point bayerization = std::chrono::steady_clock::now();

    // Debayerization of input data
    demosaic<<<numBlocks, blockSize>>>(M,N, temporary, R,G,B);
    std::chrono::steady_clock::time_point demos = std::chrono::steady_clock::now();

    // Transposing the green channel of the RGB picture and converting to a double array.
    // Transposition is necessary for the FFT to work properly
    transposeU16ToDouble<<<numBlocks, blockSize>>>(N, M, G, doubleTemporary);
    std::chrono::steady_clock::time_point transposition = std::chrono::steady_clock::now();

    // Converting the double array to a complex array of hipfftDoubleComplex type
    convertToComplex<<<numBlocks, blockSize>>>(N*M, doubleTemporary, image);
    std::chrono::steady_clock::time_point complexization = std::chrono::steady_clock::now();

    // Declaring the FFT plan
    hipfftPlan2d(&plan, M,N, HIPFFT_C2C);
    // Execute forward FFT on the green channel
    hipfftExecC2C(plan, image, image, HIPFFT_FORWARD);
    std::chrono::steady_clock::time_point fft = std::chrono::steady_clock::now();

    // Calculating the Hq matrix according to the equations in the original .m file.
    calculate<<<numBlocks, blockSize>>>(M,N, z, dx, n, lambda, Hq);
    std::chrono::steady_clock::time_point calculation = std::chrono::steady_clock::now();

    // Element-wise multiplication of Hq matrix and the image
    transposedMultiplication<<<numBlocks, blockSize>>>(N, M, Hq, image);
    std::chrono::steady_clock::time_point transMult = std::chrono::steady_clock::now();

    // Executing inverse FFT
    hipfftExecC2C(plan, image, image, HIPFFT_BACKWARD);
    // Freeing the memory of FFT plan
    hipfftDestroy(plan);
    std::chrono::steady_clock::time_point ifft = std::chrono::steady_clock::now();

    // Conversion of result matrix to a real double matrix
    absoluteValue<<<numBlocks, blockSize>>>(M,N, image, outputArray);
    std::chrono::steady_clock::time_point absolution = std::chrono::steady_clock::now();

    // Copying the memory from the outputArray to the mex Output
    // desample<<<numBlocks, blockSize>>>(M,N, outputArray, smallerOutputArray);
    hipMemcpy(mxGetPr(Erz), outputArray, N*M*sizeof(float),hipMemcpyDeviceToHost);
    std::chrono::steady_clock::time_point finalMemcpy = std::chrono::steady_clock::now();
    // Waiting for all the CUDA operations to finish (probably not necessary)
    hipDeviceSynchronize();

    // Freeing memory
    hipFree(doubleComplexArray);
    hipFree(doubleArray);
    hipFree(input);
    hipFree(RGB);
    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();


    /*
    * Printing the time measurements
    */
    mexPrintf("Setup took %d us\n", std::chrono::duration_cast<std::chrono::microseconds>(setup - start).count());
    mexPrintf("Allocating memory took %d us\n", std::chrono::duration_cast<std::chrono::microseconds>(mallocs - setup).count());
    mexPrintf("Bayerizing took %d us\n", std::chrono::duration_cast<std::chrono::microseconds>(bayerization - mallocs).count());
    mexPrintf("Demosaic took %d us\n", std::chrono::duration_cast<std::chrono::microseconds>(demos - bayerization).count());
    mexPrintf("Transposition of raw data took %d us\n", std::chrono::duration_cast<std::chrono::microseconds>(transposition - demos).count());
    mexPrintf("Conversion to complex data took %d us\n", std::chrono::duration_cast<std::chrono::microseconds>(complexization - transposition).count());
    mexPrintf("FFT took %d us\n", std::chrono::duration_cast<std::chrono::microseconds>(fft - complexization).count());
    mexPrintf("Calculation of Hq took %d us\n", std::chrono::duration_cast<std::chrono::microseconds>(calculation - fft).count());
    mexPrintf("Multiplication took %d us\n", std::chrono::duration_cast<std::chrono::microseconds>(transMult - calculation).count());
    mexPrintf("Inverse FFT took %d us\n", std::chrono::duration_cast<std::chrono::microseconds>(ifft - transMult).count());
    mexPrintf("Absolute Value %d us\n", std::chrono::duration_cast<std::chrono::microseconds>(absolution - ifft).count());
    mexPrintf("Final memcpy took %d us\n", std::chrono::duration_cast<std::chrono::microseconds>(finalMemcpy - absolution).count());
    mexPrintf("Final memory emptying took %d us\n", std::chrono::duration_cast<std::chrono::microseconds>(end - finalMemcpy).count());
    mexPrintf("All together it took %d us\n\n", std::chrono::duration_cast<std::chrono::microseconds>(end - start).count());
}