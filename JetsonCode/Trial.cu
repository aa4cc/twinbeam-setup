#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "cudaEGL.h"
#include "cuda_egl_interop.h"
#include "Argus/Argus.h"
#include "EGLStream/EGLStream.h"
#include "stdio.h"
#include "stdlib.h"
#include "EGL/egl.h"
#include <iterator>
#include <unistd.h>
#include <iostream>
#include <fstream>
#include <chrono>
#include <opencv2/opencv.hpp>
#include <opencv2/highgui/highgui.hpp>
#include "Kernels.h"
#include <cstdlib>
#include <thread>
#include <mutex>
#include <sys/socket.h>
#include <netinet/in.h>
#include <arpa/inet.h>
#include <string.h>
#include "Definitions.h"
#include "cxxopts.hpp"

#define BUFSIZE 1000
#define PORT 30000

static const int    DEFAULT_FPS        = 30;

using namespace std;
using namespace Argus;
using namespace EGLStream;

hipError_t res;

float* doubleArray;
float* outputArray;
float* convoOutputArray;
float* convoOutputArrayRed;
hipfftComplex* kernelGreen;
hipfftComplex* kernelRed;

float* redConverted;

float* convolutionMaskGreen;
float* convolutionMaskRed;
float* convoOutputArrayGreen;

float* maximaRed;
float* maximaGreen;
float* doubleTemporary;

int* greenPoints;
int* redPoints;
int* positionsGreen;
int* positionsRed;

int* redPointsLast;
int* greenPointsLast;
int* current_index;

bool playSequence;
bool sleeping;
bool connected;
bool initialized;
bool requested_image;
REQUEST_TYPE requested_type;
bool send_points;
bool force_exit;
bool touch_kill;
int settings[7];
int client;

// Options
bool opt_verbose	= false;
bool opt_debug		= false;
bool opt_show		= false;

uint16_t *R;
uint16_t *G;

mutex mtx;
mutex outputMtx;

int numBlocks;

bool quitSequence;
short cycles;

std::chrono::duration<double> elapsed_seconds_average;
std::chrono::duration<double> initialization_seconds_average;
std::chrono::duration<double> conversion_seconds_average;
std::chrono::duration<double> back_propagation_seconds_average;
std::chrono::duration<double> convolution_seconds_average;
std::chrono::duration<double> localmaxima_seconds_average;
std::chrono::duration<double> sorting_seconds_average;


EGLStreamKHR eglStream;

const textureReference* uvTex;
const textureReference* yTex;

texture<unsigned char, 2, hipReadModeElementType> yTexRef;
texture<uchar2, 2, hipReadModeElementType> uvTexRef;

struct is_zero
{
  __host__ __device__
  bool operator()(const int &x)
  {
    return (x == 0);
  }
};


// cxxopts.hpp related definitions
cxxopts::ParseResult
parse(int argc, char* argv[])
{
  try
  {
    cxxopts::Options options(argv[0], " - Twin-beam setup - image processing");
    options
      .positional_help("[optional args]")
      .show_positional_help();

    options
      .add_options()
      ("s,show", "Display the processed image on the display", 	cxxopts::value<bool>(opt_show))
      ("d,debug", "Prints debug information",					cxxopts::value<bool>(opt_debug))
      ("v,verbose", "Prints some additional information",		cxxopts::value<bool>(opt_verbose))
      ("help", "Prints help")
    ;

    auto result = options.parse(argc, argv);

    if (result.count("help"))
    {
      std::cout << options.help({"", "Group"}) << std::endl;
      exit(0);
    }


    if (opt_debug) {
	    if (opt_show)
	    {
	      std::cout << "Saw option ‘s’" << std::endl;
	    }

	    if (opt_debug)
	    {
	      std::cout << "Saw option ‘d’" << std::endl;
	    }

	    if (opt_verbose)
	    {
	      std::cout << "Saw option ‘v’" << std::endl;
	    }
	}


    return result;

  } catch (const cxxopts::OptionException& e)
  {
    std::cout << "error parsing options: " << e.what() << std::endl;
    exit(1);
  }
}


int processPoints(float* inputPoints, int* outputArray){
	int* positions;
	int* positionsSorted;
	int* counter;
	float* points;
	int* counting;
	int* temp;
	
	hipMalloc(&points, settings[0]*settings[1]*sizeof(float));
	hipMalloc(&positions, settings[0]*settings[1]*sizeof(int));
	hipMalloc(&positionsSorted, settings[0]*settings[1]*sizeof(int));
	hipMalloc(&counter, sizeof(int));
	hipHostMalloc(&counting,sizeof(int));
    memset(counting, 0, sizeof(int));
	
	hipMemcpy(points, inputPoints, sizeof(float)*settings[0]*settings[1], hipMemcpyDeviceToDevice);
	findPoints<<<numBlocks, BLOCKSIZE>>>(settings[0], settings[1], points, positions, counter);
	stupidSort<<<numBlocks, BLOCKSIZE>>>(settings[0], settings[1],positions, positionsSorted, counting);
	hipMemcpy(counting, counter, sizeof(int), hipMemcpyDeviceToHost);
	temp = (int*)malloc(sizeof(int)*counting[0]);
	hipMemcpy(temp, positionsSorted, sizeof(int)*counting[0], hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipFree(points);
	hipFree(positions);
	hipFree(counter);
	hipFree(positionsSorted);
	outputArray = temp;
	return counting[0];
}

//#region

__global__ void yuv2bgr(int width, int height, int offset_x, int offset_y,
						uint16_t* G, uint16_t* R)
        {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            int count = width*height;
            int tx, ty, ty2;
            float y1, y2;
            float u1, v2, v1;
            for (int i = index; i < count; i += stride)
            {
            	ty = i/width + offset_y;
            	ty2 = i/width + offset_y - (512-50);
            	tx = i%width + offset_x;
            	y1 = (float)((tex2D<unsigned char>(yTexRef, (float)tx+0.5f, (float)ty+0.5f) - (float)16) * 1.164383f);
            	y2 = (float)((tex2D<unsigned char>(yTexRef, (float)tx+0.5f, (float)ty2+0.5f) - (float)16) * 1.164383f);
            	u1 = (float)(tex2D<uchar2>(uvTexRef, (float)(tx/2)+(float)(tx%2)+0.5f,
            	 	 (float)(ty/2)+(float)(ty%2)+0.5f).x - 128) * 0.391762f;
            	v2 = (float)(tex2D<uchar2>(uvTexRef, (float)(tx/2)+(float)(tx%2)+0.5f,
            	     (float)(ty2/2)+(float)(ty2%2)+0.5f).y - 128) * 1.596027f;
            	v1 = (float)(tex2D<uchar2>(uvTexRef, (float)(tx/2)+(float)(tx%2)+0.5f,
            	     (float)(ty/2)+(float)(ty%2)+0.5f).y - 128) * 0.812968f;
				G[i] = (uint16_t)(y1-u1-v1);
				R[i] = (uint16_t)(y2+v2+u1/10);
            }
        }

void transformKernel(int M, int N, int kernelDim, float* kernel, hipfftComplex* outputKernel){
	
	kernelToImage<<<numBlocks, BLOCKSIZE>>>(M, N, kernelDim, kernel, outputKernel);
   
    hipfftHandle plan;
    
    hipfftPlan2d(&plan, N,M, HIPFFT_C2C);
    
    hipfftExecC2C(plan, outputKernel, outputKernel, HIPFFT_FORWARD);
    
    hipfftDestroy(plan);
}

void printArray(float* array, int width, int height){
	for(int i = 0; i < height; i++){
		for (int j = 0; j < width; j++){
			printf("%f ", array[j + height*i]);
			}
			printf("\n");
		}
}


void h_backPropagate(int M, int N, float lambda, float z, float* input,
		hipfftComplex* kernel, float* output, float* output2, bool display)
{
    hipfftComplex* doubleComplexArray;
    hipfftComplex* Hq;
    hipfftComplex* image;
    hipfftComplex* kernelizedImage;
    float* temporary;
    float* extremes;
    hipfftHandle plan;

    hipMalloc(&doubleComplexArray, 3*N*M*sizeof(hipfftComplex));
    Hq = &doubleComplexArray[0];
    image = &doubleComplexArray[N*M];
    kernelizedImage = &doubleComplexArray[2*N*M];
    hipMalloc(&temporary, N*M*sizeof(float));
    hipMalloc(&extremes, sizeof(float));

    convertToComplex<<<numBlocks, BLOCKSIZE>>>(N*M, input, image);
    // Declaring the FFT plan
    hipfftPlan2d(&plan, N,M, HIPFFT_C2C);
    // Execute forward FFT on the green channel
    hipfftExecC2C(plan, image, image, HIPFFT_FORWARD);
    // Calculating the Hq matrix according to the equations in the original .m file.
    calculate<<<numBlocks, BLOCKSIZE>>>(N,M, z, PIXEL_DX, REFRACTION_INDEX, lambda, Hq);
    // Element-wise multiplication of Hq matrix and the image
	elMultiplication<<<numBlocks, BLOCKSIZE>>>(M, N, Hq, image);
	elMultiplication2<<<numBlocks, BLOCKSIZE>>>(M, N, image, kernel, kernelizedImage);
    if(display){
		// Executing inverse FFT
		hipfftExecC2C(plan, image, image, HIPFFT_BACKWARD);
		// Conversion of result matrix to a real double matrix
		absoluteValue<<<numBlocks, BLOCKSIZE>>>(M,N, image, output);

		findExtremes<<<numBlocks, BLOCKSIZE>>>(M,N, output, extremes);
		normalize<<<numBlocks, BLOCKSIZE>>>(M,N, output, extremes);
	}
	hipfftExecC2C(plan, kernelizedImage, kernelizedImage, HIPFFT_BACKWARD);
	cutAndConvert<<<numBlocks, BLOCKSIZE>>>(M,N,kernelizedImage, convoOutputArrayGreen);
    hipFree(extremes);
    hipMalloc(&extremes, sizeof(float));
	findExtremes<<<numBlocks, BLOCKSIZE>>>(M,N, convoOutputArrayGreen, extremes);
	normalize<<<numBlocks, BLOCKSIZE>>>(M,N, convoOutputArrayGreen, extremes);
	getLocalMaxima<<<numBlocks, BLOCKSIZE>>>(M,N,convoOutputArrayGreen,output2);
	// Freeing the memory of FFT plan
	hipfftDestroy(plan);

    hipFree(extremes);
    hipFree(doubleComplexArray);
    hipFree(temporary);
}

void printErrorRuntime(hipError_t result){
	const char* pstr = hipGetErrorName(result);
	//printf("%s\n", pstr);
}


void changeSettings(char* buf){
	int tmpSettings; 
	int count = 0;
	int current_index = 2;
	while(count < 7){
		string str = "";
		while(isdigit(buf[current_index])){
			str.append(1u,buf[current_index]);
			current_index++;
		}
		try{
			tmpSettings = atol(str.c_str());
		}
		catch(int e ){
			printf("Number is too large\n");
			tmpSettings = 0;
		}
		if(tmpSettings != 0){
			settings[count] = tmpSettings;
			printf("%d\n", settings[count]);
		}
		count++;
		current_index++;
	}
}

MESSAGE_TYPE parseMessage(char* buf){
		switch (buf[0]){
			case 's':
				return MSG_WAKEUP;
			case 'q':
				return MSG_SLEEP;
			case 'o':
				return MSG_SETTINGS;
			case 'a':
				return MSG_HELLO;
			case 'd':
				return MSG_DISCONNECT;
			case 'r':
				requested_type = BACKPROPAGATED;
				return MSG_REQUEST;
			case 'x':
				return MSG_REQUEST_RAW_G;
			case 'y':
				return MSG_REQUEST_RAW_R;
			default:
				return MSG_UNKNOWN_TYPE;
		}
}



void keyboard_thread(){
	printf("keyboard_thread: started\n");

	char input;
	while(!force_exit){
		input = getchar();
		if(input == 's'){
			printf("Putting the process to sleep\n");
			sleeping = true;
			initialized = false;
		}
		else if(input == 'c'){
			printf("Connected the main manipulation computer\n");
			connected = true;
		}
		else if(input == 'w'){
			printf("Starting the program from keyboard\n");
			initialized = true;
			sleeping = false;
		}
		else if(input == 'd'){
			connected = false;
			sleeping = true;
			initialized = false;
		}
		else if(input == 'e'){
			force_exit = true;
		}		
	}

	printf("keyboard_thread: ended\n");
}

void input_thread(){
	printf("input_thread: started\n");

	std::string text;
	sockaddr_in sockName;
	sockaddr_in clientInfo; 
	int mainSocket;
	char buf[BUFSIZE];
	socklen_t addrlen;
	MESSAGE_TYPE response;
	int size;
	
	mainSocket = socket(AF_INET, SOCK_STREAM, IPPROTO_TCP);
	if(mainSocket == -1)
		printf("Couldn't create socket!\n");
	sockName.sin_family = AF_INET;
	sockName.sin_port =	htons(PORT);
	sockName.sin_addr.s_addr = INADDR_ANY;

	int yes = 1;
	if ( setsockopt(mainSocket, SOL_SOCKET, SO_REUSEADDR, &yes, sizeof(int)) == -1 )
	{
	    perror("setsockopt");
	}

	bind(mainSocket, (sockaddr*)&sockName, sizeof(sockName));
	listen(mainSocket, 10000000);
	while(!force_exit){
		
		addrlen = sizeof(clientInfo);
		client = accept(mainSocket, (sockaddr*)&clientInfo, &addrlen);
		cout << "Got a connection from " << inet_ntoa((in_addr)clientInfo.sin_addr) << endl;
		if (client != -1)
		 {
			 connected = true;
		 }

		while(connected && !force_exit){
			int msg_len = recv(client, buf, BUFSIZE - 1, 0);

			if (msg_len == -1)
			{
				printf("Error while receiving data\n");
			}

			printf("Received bytes: %d\n", msg_len);

			response = parseMessage(buf);
			switch(response){
				case MSG_WAKEUP:
				{
					sleeping = false;
					initialized = true;
					break;
				}
				case MSG_SLEEP:
				{
					sleeping = true;
					initialized = false;
					break;
				}
				case MSG_SETTINGS:
				{
					if(sleeping == false)
						printf("Can't change settings while the loop is running\n");
					else{
						changeSettings(buf);
						printf("Changed settings\n");
					}
					break;
				}
				case MSG_DISCONNECT:
				{
					connected = false;
					sleeping = true;
					initialized = false;
					break;
				}
				case MSG_REQUEST:
					requested_image = true;
					break;
				case MSG_REQUEST_RAW_G:
					requested_image = true;
					requested_type = RAW_G;
					break;
				case MSG_REQUEST_RAW_R:
					requested_image = true;
					requested_type = RAW_R;
					break;
				case MSG_HELLO:
				{
					send(client, "Hello!",7,0);
					break;
				}	
			} 
		}
		close(client);
	}
	close(mainSocket);

	printf("input_thread: ended\n");
}

//#end_region
void consumer_thread(){
	printf("consumer_thread: started\n");
	//Initializing LibArgus according to the tutorial for a sample project.
	// First we create a CameraProvider, necessary for each project.
	UniqueObj<CameraProvider> cameraProvider(CameraProvider::create());
	ICameraProvider* iCameraProvider = interface_cast<ICameraProvider>(cameraProvider);
	if(!iCameraProvider){
		printf("Failed to establish libargus connection\n");
	}
	
	// Second we select a device from which to receive pictures (camera)
	std::vector<CameraDevice*> cameraDevices;
	iCameraProvider->getCameraDevices(&cameraDevices);
	if (cameraDevices.size() == 0){
		printf("No camera devices available\n");
	}
	CameraDevice *selectedDevice = cameraDevices[0];

	// We create a capture session 
	UniqueObj<CaptureSession> captureSession(iCameraProvider->createCaptureSession(selectedDevice));
	ICaptureSession *iCaptureSession = interface_cast<ICaptureSession>(captureSession);
	if (!iCaptureSession){
 		printf("Failed to create CaptureSession\n");
	}
	
	//CUDA variable declarations
	cudaEglStreamConnection conn;
	hipGraphicsResource_t resource;
	cudaEglFrame eglFrame;		
	hipArray_t yArray;
	hipArray_t uvArray;
	float* extremesMapGreen;
	
	
	
	hipChannelFormatDesc yChannelDesc;
	hipChannelFormatDesc uvChannelDesc;
	while(!force_exit){
		while(connected && !force_exit){
			while(sleeping && connected && !force_exit){}
			if (force_exit) break;
			// Managing the settings for the capture session.
			UniqueObj<OutputStreamSettings> streamSettings(iCaptureSession->createOutputStreamSettings());
			IOutputStreamSettings *iStreamSettings = interface_cast<IOutputStreamSettings>(streamSettings);
			iStreamSettings->setPixelFormat(PIXEL_FMT_YCbCr_420_888);
			iStreamSettings->setResolution(Size2D<uint32_t>(WIDTH,HEIGHT));
			
			// Creating an Output stream. This should already create a producer.
			UniqueObj<OutputStream> outputStream(iCaptureSession->createOutputStream(streamSettings.get()));
			IStream* iStream = interface_cast<IStream>(outputStream);
			if (!iStream){
				printf("Failed to create OutputStream\n");
			}
			eglStream = iStream->getEGLStream();
			cudaEGLStreamConsumerConnect(&conn, eglStream);
			
			// Managing requests.
			UniqueObj<Request> request(iCaptureSession->createRequest());
			IRequest *iRequest = interface_cast<IRequest>(request);
			iRequest->enableOutputStream(outputStream.get());
			
			ISourceSettings *iSourceSettings = interface_cast<ISourceSettings>(iRequest->getSourceSettings());
			iSourceSettings->setFrameDurationRange(Range<uint64_t>(1e9/DEFAULT_FPS));
			iSourceSettings->setExposureTimeRange(Range<uint64_t>(settings[4]));

			hipMalloc(&G, settings[0]*settings[1]*sizeof(uint16_t));
			hipMalloc(&R, settings[0]*settings[1]*sizeof(uint16_t));
			hipMalloc(&positionsGreen, settings[0]*settings[1]*sizeof(float));
			hipMalloc(&positionsRed, settings[0]*settings[1]*sizeof(float));
			
			
			hipMalloc(&convolutionMaskGreen, CONVO_DIM_GREEN*CONVO_DIM_GREEN*sizeof(float));
			hipMalloc(&convolutionMaskRed, CONVO_DIM_RED*CONVO_DIM_RED*sizeof(float));
			numBlocks = 1024;
			generateConvoMaskGreen<<<numBlocks, BLOCKSIZE>>>(CONVO_DIM_GREEN, CONVO_DIM_GREEN, convolutionMaskGreen);
			generateConvoMaskRed<<<numBlocks, BLOCKSIZE>>>(CONVO_DIM_RED, CONVO_DIM_RED, convolutionMaskRed);
			
			hipMalloc(&kernelGreen, settings[0]*settings[1]*sizeof(hipfftComplex));
			hipMalloc(&kernelRed, settings[0]*settings[1]*sizeof(hipfftComplex));
			
			transformKernel(settings[0], settings[1], CONVO_DIM_GREEN, convolutionMaskGreen, kernelGreen);
			transformKernel(settings[0], settings[1], CONVO_DIM_RED, convolutionMaskRed, kernelRed);
			
			hipMalloc(&convoOutputArrayGreen, settings[0]*settings[1]*sizeof(float));
			hipMalloc(&convoOutputArrayRed, settings[0]*settings[1]*sizeof(float));
			hipMallocManaged(&current_index, sizeof(int));
			mtx.lock();
			hipMalloc(&maximaGreen, settings[0]*settings[1]*sizeof(float));
			hipMalloc(&maximaRed, settings[0]*settings[1]*sizeof(float));
			hipMalloc(&doubleArray, 2*settings[1]*settings[0]*sizeof(float));
			doubleTemporary = &doubleArray[0];
			outputArray = &doubleArray[settings[1]*settings[0]];
			hipMalloc(&convoOutputArray, settings[0]*settings[1]*sizeof(float));
			hipMalloc(&redConverted, settings[0]*settings[1]*sizeof(float));
			mtx.unlock();

			yTexRef.normalized = 0;
			yTexRef.filterMode = hipFilterModePoint;
			yTexRef.addressMode[0] = hipAddressModeClamp;
			yTexRef.addressMode[1] = hipAddressModeClamp;
			hipGetTextureReference(&yTex, HIP_SYMBOL(&yTexRef));
			
			uvTexRef.normalized = 0;
			uvTexRef.filterMode = hipFilterModePoint;
			uvTexRef.addressMode[0] = hipAddressModeClamp;
			uvTexRef.addressMode[1] = hipAddressModeClamp;
			hipGetTextureReference(&uvTex, HIP_SYMBOL(&uvTexRef));
			
			//CUDA initialization
			//Main loop
			auto initializer = std::chrono::system_clock::now();
			std::chrono::duration<double> elapsed_seconds_average = initializer-initializer;
			std::chrono::duration<double> initialization_seconds_average = initializer-initializer;
			std::chrono::duration<double> conversion_seconds_average = initializer-initializer;
			std::chrono::duration<double> back_propagation_seconds_average = initializer-initializer;
			std::chrono::duration<double> convolution_seconds_average = initializer-initializer;
			std::chrono::duration<double> localmaxima_seconds_average = initializer-initializer;
			std::chrono::duration<double> sorting_seconds_average = initializer-initializer;

			int final_count = 0;
			while(!initialized && connected && !force_exit){}
			if (force_exit) break;

			while(!sleeping && connected && ! force_exit){
				auto start = std::chrono::system_clock::now();
				
				
				iCaptureSession->capture(request.get());
				res = cudaEGLStreamConsumerAcquireFrame(&conn, &resource, 0, 5000);
				if(res != hipSuccess){
					continue;
				}
				cudaGraphicsResourceGetMappedEglFrame(&eglFrame, resource, 0, 0);
				yArray = eglFrame.frame.pArray[0];
				uvArray = eglFrame.frame.pArray[1];
				
				hipGetChannelDesc(&yChannelDesc, (hipArray_const_t)(yArray));
				hipBindTextureToArray(yTex, (hipArray_const_t)(yArray), &yChannelDesc);
				hipGetChannelDesc(&uvChannelDesc, (hipArray_const_t)(uvArray));
				hipBindTextureToArray(uvTex, (hipArray_const_t)(uvArray), &uvChannelDesc);
				auto initialization = std::chrono::system_clock::now();

				
				numBlocks = (settings[0]*settings[1]/2 +BLOCKSIZE -1)/BLOCKSIZE;
				yuv2bgr<<<numBlocks, BLOCKSIZE>>>(settings[0], settings[1], settings[2], settings[3], G, R);
				auto test = std::chrono::system_clock::now();
				conversion_seconds_average += test - initialization;
				initialization_seconds_average += initialization-start;
				u16ToDouble<<<numBlocks, BLOCKSIZE>>>(settings[0], settings[1], G, doubleTemporary);
				u16ToDouble<<<numBlocks, BLOCKSIZE>>>(settings[0], settings[1], R, redConverted);
				mtx.lock();
				h_backPropagate(settings[0], settings[1], LAMBDA_GREEN, (float)settings[6]/(float)1000000,
						doubleTemporary, kernelGreen, outputArray, maximaGreen, true);		
				h_backPropagate(settings[0],settings[1], LAMBDA_RED, (float)settings[5]/(float)1000000,
						redConverted, kernelRed, convoOutputArray, maximaRed, false);
				mtx.unlock();
				
				
				auto test2 = std::chrono::system_clock::now();
				std::chrono::duration<double> elapsed_seconds = test2-test;
				
				if(opt_verbose) {
					std::cout << "Converting the image format took: " << elapsed_seconds.count() << "s\n";
				}

				hipUnbindTexture(yTex);
				hipUnbindTexture(uvTex);
				
				cudaEGLStreamConsumerReleaseFrame(&conn, resource, 0);
				
				auto end = std::chrono::system_clock::now();
				elapsed_seconds = end-start;
				elapsed_seconds_average +=elapsed_seconds;
				final_count++;
				send_points = true;
				
				if(opt_verbose) {
					std::cout << "This cycle took: " << elapsed_seconds.count() << "s\n";
					printf("%d\n", quitSequence);
				}

				cycles++;				
			}
			std::cout << "average complete: " << elapsed_seconds_average.count()/final_count << "s\n";
			iCaptureSession->waitForIdle();
			
			hipFree(G);
			hipFree(R);
			
			hipFree(doubleArray);
			hipFree(convoOutputArray);
			hipFree(redConverted);
			hipFree(maximaGreen);
			hipFree(maximaRed);
			hipFree(convolutionMaskGreen);
			hipFree(convolutionMaskRed);
			hipFree(convoOutputArrayRed);
			hipFree(kernelGreen);
			hipFree(kernelRed);
			
			cudaEGLStreamConsumerDisconnect(&conn);
			iStream->disconnect();
			outputStream.reset();
		}
	}

	printf("consumer_thread: ended\n");
}

void CallBackFunc(int event, int x, int y, int flags, void* userdata)
{
	// https://www.opencv-srf.com/2011/11/mouse-events.html
	if ( event == CV_EVENT_MOUSEMOVE )
     {
        cout << "Mouse move over the window - position (" << x << ", " << y << ")" << endl;
        if (touch_kill) {
        	force_exit = true;
      	}
     }
}

void print_thread(){
	printf("print_thread: started\n");

	float* tempArray;
	float* tempArray2;
	while(true){
		while(sleeping && connected && !force_exit){}
		if (force_exit) break;
			
		hipMalloc(&tempArray, sizeof(float)*settings[0]*settings[1]);
		hipMalloc(&tempArray2, sizeof(float)*settings[0]*settings[1]);
		float* output = (float*)malloc(sizeof(float)*settings[0]*settings[1]);
		float* output2 = (float*)malloc(sizeof(float)*settings[0]*settings[1]);
		cv::namedWindow("Basic Visualization", CV_WINDOW_NORMAL);
		cv::setWindowProperty("Basic Visualization", CV_WND_PROP_FULLSCREEN, CV_WINDOW_FULLSCREEN);
		//set the callback function for any mouse event
     	cv::setMouseCallback("Basic Visualization", CallBackFunc, NULL);

		while(!initialized && connected && !force_exit){}
		if (force_exit) break;

		while(!sleeping && connected){
			if(cycles >= 3){
				cycles = 0;
				mtx.lock();
				hipMemcpy(tempArray, maximaGreen, sizeof(float)*settings[0]*settings[1], hipMemcpyDeviceToDevice);
				hipMemcpy(tempArray2, outputArray, sizeof(float)*settings[0]*settings[1], hipMemcpyDeviceToDevice);
				mtx.unlock();
				hipMemcpy(output, tempArray, sizeof(float)*settings[0]*settings[1], hipMemcpyDeviceToHost);
				hipMemcpy(output2, tempArray2, sizeof(float)*settings[0]*settings[1], hipMemcpyDeviceToHost);
				const cv::Mat img(cv::Size(settings[0], settings[1]), CV_32F, output);
				const cv::Mat img2(cv::Size(settings[0], settings[1]), CV_32F, output2);

				const cv::Mat img2_trans(cv::Size(settings[0], settings[1]), CV_32F);

				cv::flip(img2, img2_trans, -1);
				cv::transpose(img2_trans, img2);

				const cv::Mat result = img2;
				cv::imshow("Basic Visualization", result);
				cv::waitKey(1);
			}
			else{
				usleep(5000);
			}

			if (force_exit) break;
		}
		hipFree(tempArray);
		hipFree(tempArray2);
		free(output);
		free(output2);
	}

	printf("print_thread: ended\n");
}


void output_thread(){
	printf("output_thread: started\n");

	float *temporary;
	float *temporary_red_positions;
	float *temporary_green_positions;
	int *sorted_red_positions;
	int *sorted_green_positions;
	while(true){
		while(sleeping && !force_exit){}
		if (force_exit) break;

		hipMalloc(&temporary, settings[0]*settings[1]*sizeof(float));
		hipMalloc(&temporary_red_positions, settings[0]*settings[1]*sizeof(float));
		hipMalloc(&temporary_green_positions, settings[0]*settings[1]*sizeof(float));
		
		char* buffer = (char*)malloc(settings[0]*settings[1]*sizeof(float));
		
		while(connected && !sleeping){
			if(requested_image){
				mtx.lock();
				switch (requested_type){
					case BACKPROPAGATED:
						hipMemcpy(temporary, outputArray, sizeof(float)*settings[0]*settings[1], hipMemcpyDeviceToDevice);
						break;
					case RAW_G:
						hipMemcpy(temporary, doubleTemporary, sizeof(float)*settings[0]*settings[1], hipMemcpyDeviceToDevice);
						break;
					case RAW_R:
						hipMemcpy(temporary, redConverted, sizeof(float)*settings[0]*settings[1], hipMemcpyDeviceToDevice);
						break;
				}	

				mtx.unlock();
				hipMemcpy(buffer, temporary, sizeof(float)*settings[0]*settings[1], hipMemcpyDeviceToHost);
				send(client, buffer, sizeof(float)*settings[0]*settings[1], 0);
				printf("Image sent!\n");
				requested_image = false;
			}
			if(send_points){
				auto test3 = std::chrono::system_clock::now();
				outputMtx.lock();
				hipMemcpy(temporary_green_positions, maximaGreen, sizeof(int)*settings[0]*settings[1], hipMemcpyDeviceToHost);
				hipMemcpy(temporary_red_positions, maximaGreen, sizeof(int)*settings[0]*settings[1], hipMemcpyDeviceToHost);
				outputMtx.unlock();
				int greenCount = processPoints(temporary_green_positions, sorted_green_positions);
				int redCount = processPoints(temporary_red_positions, sorted_red_positions);
				//send(client, buffer, sizeof(float)*settings[0]*settings[1], 0);
				//send(client, buffer, sizeof(float)*settings[0]*settings[1], 0);
				// printf("%d; %d\n", redCount, greenCount);
				send_points = false;
				auto test4 = std::chrono::system_clock::now();
				sorting_seconds_average += test4-test3;
			}

			usleep(1000);

			if (force_exit) break;
		}
		hipFree(temporary_red_positions);
		hipFree(temporary_green_positions);
		
		hipFree(temporary);
		free(buffer);		
	}

	printf("output_thread: ended\n");
}

int main(int argc, char* argv[]){
	// cxxopt.h initialization
  	auto result = parse(argc, argv);

	force_exit = false;
	
	cycles = 0;
	settings[0] = 1024;
	settings[1] = 1024;
	settings[2] = 1195;
	settings[3] = 500;
	settings[4] = 5000000;
	settings[5] = 3100;
	settings[6] = 2400;
	
	quitSequence = false;
	playSequence = false;
	send_points = false;

	if (opt_show) {
		initialized = true;
		connected = true;
		sleeping = false;
		touch_kill = true;
	} else {
		connected = false;
		sleeping = true;
		touch_kill = false;
	}

	thread consumr_thr (consumer_thread);
	thread print_thr (print_thread);
	thread input_thr (input_thread);
	thread output_thr (output_thread);
	thread keyboard_thr (keyboard_thread);
	
	consumr_thr.join();
	print_thr.join();
	// input_thr.join();
	output_thr.join();
	// keyboard_thr.join();


	return 0;
}
