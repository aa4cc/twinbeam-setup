#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "stdio.h"
#include "stdlib.h"
#include <iterator>
#include <unistd.h>
#include <iostream>
#include <fstream>
#include <chrono>
#include <opencv2/opencv.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/core/cuda.hpp>
#include "Kernels.h"
#include <cstdlib>
#include <thread>
#include <sys/socket.h>
#include <netinet/in.h>
#include <arpa/inet.h>
#include <string.h>
#include "Definitions.h"
#include "Misc.h"
#include "Settings.h"
#include "argpars.h"
#include "camera_thread.h"
#include "BackPropagator.h"
#include "BeadsFinder.h"
#include "ImageData.h"

using namespace std;

ImageData<uint8_t> G, R, G_backprop;
uint16_t bead_positions[2*BeadsFinder::MAX_NUMBER_BEADS];
uint32_t bead_count;

int client;

int img_count = 0;
std::chrono::duration<double> elapsed_seconds_average;


void keyboard_thread(){
	printf("INFO: keyboard_thread: started\n");

	char input;
	while(!Settings::force_exit){
		input = getchar();
		if(input == 's'){
			printf("INFO: Putting the process to sleep.\n");
			Settings::set_sleeping(true);
			Settings::set_initialized(false);
		}
		else if(input == 'c'){
			printf("INFO: Simulating connection to main computation unit.\n");
			Settings::set_connected(true);
		}
		else if(input == 'w'){
			printf("INFO: Starting the program from keyboard.\n");
			Settings::set_initialized(true);
			Settings::set_sleeping(false);
		}
		else if(input == 'd'){
			Settings::set_connected(false);
			Settings::set_sleeping(true);
			Settings::set_initialized(false);
		}
		else if(input == 'e'){
			Settings::set_force_exit(true);
		}		
	}

	printf("INFO: keyboard_thread: ended\n");
}

void network_thread(){
	printf("INFO: network_thread: started\n");

	std::string text;
	sockaddr_in sockName;
	sockaddr_in clientInfo; 
	int mainSocket;
	char buf[BUFSIZE];
	socklen_t addrlen;
	MESSAGE_TYPE response;
	
	mainSocket = socket(AF_INET, SOCK_STREAM, IPPROTO_TCP);
	if(mainSocket == -1)
		printf("ERROR: Couldn't create socket!\n");
	sockName.sin_family = AF_INET;
	sockName.sin_port =	htons(PORT);
	sockName.sin_addr.s_addr = INADDR_ANY;

	int yes = 1;
	if ( setsockopt(mainSocket, SOL_SOCKET, SO_REUSEADDR, &yes, sizeof(int)) == -1 )
	{
	    perror("setsockopt");
	}

	bind(mainSocket, (sockaddr*)&sockName, sizeof(sockName));
	listen(mainSocket, 10000000);
	while(!Settings::force_exit){
		
		addrlen = sizeof(clientInfo);
		client = accept(mainSocket, (sockaddr*)&clientInfo, &addrlen);
		cout << "INFO: Got a connection from " << inet_ntoa((in_addr)clientInfo.sin_addr) << endl;
		if (client != -1)
		 {
			 Settings::set_connected(true);
		 }

		while(Settings::connected && !Settings::force_exit){
			int msg_len = recv(client, buf, BUFSIZE - 1, 0);

			if (msg_len == -1)
			{
				printf("ERROR: Did not properly receive data.\n");
			}

			if(Options::debug)
				printf("DEBUG: Received %d bytes.\n", msg_len);

			response = parseMessage(buf);
			switch(response){
				case MSG_WAKEUP:
				{
					Settings::set_sleeping(false);
					Settings::set_initialized(true);
					break;
				}
				case MSG_SLEEP:
				{
					Settings::set_sleeping(true);
					Settings::set_initialized(false);
					break;
				}
				case MSG_SETTINGS:
				{
					if(!Settings::sleeping)
						printf("WARN: Can't change settings while the loop is running\n");
					else{
						memcpy(Settings::values, buf+1, sizeof(uint32_t)*STG_NUMBER_OF_SETTINGS);
						Settings::print();
						printf("INFO: Changed settings\n");
					}
					break;
				}
				case MSG_DISCONNECT:
				{
					Settings::set_connected(false);
					Settings::set_sleeping(true);
					Settings::set_initialized(false);
					break;
				}
				case MSG_REQUEST:
					Settings::set_requested_image(true);
					Settings::set_requested_type(BACKPROPAGATED);
					break;
				case MSG_REQUEST_RAW_G:
					Settings::set_requested_image(true);
					Settings::set_requested_type(RAW_G);
					break;
				case MSG_REQUEST_RAW_R:
					Settings::set_requested_image(true);
					Settings::set_requested_type(RAW_R);
					break;
				case MSG_COORDS:
					Settings::set_requested_coords(true);
					break;
				case MSG_HELLO:
				{
					send(client, "Hello!",7,0);
					break;
				}	
			} 
		}
		close(client);
	}
	close(mainSocket);

	printf("INFO: network_thread: ended\n");
}

void mouseEventCallback(int event, int x, int y, int flags, void* userdata)
{
	// https://www.opencv-srf.com/2011/11/mouse-events.html
	if ( event == cv::EVENT_MOUSEMOVE )
     {
     	if(Options::debug)
        	cout << "DEBUG: Mouse move over the window - position (" << x << ", " << y << ")" << endl;
        Settings::set_force_exit(true);
     }
}

void datasend_thread(){
	printf("INFO: datasend_thread: started\n");

	while(true){
		while(Settings::sleeping && !Settings::force_exit){}
		if (Settings::force_exit) break;
		
		while(Settings::connected && !Settings::sleeping){
			if(Settings::requested_image){
				ImageData<uint8_t> temp_img(Settings::values[STG_WIDTH], Settings::values[STG_HEIGHT]);

				switch (Settings::requested_type){
					case BACKPROPAGATED:
						G_backprop.copyTo(temp_img);
						break;
					case RAW_G:
						G.copyTo(temp_img);
						break;
					case RAW_R:
						R.copyTo(temp_img);
						break;
				}	

				send(client, temp_img.hostPtr(), sizeof(uint8_t)*Settings::get_area(), 0);
				printf("INFO: Image sent.\n");
				Settings::set_requested_image(false);
			}

			// if(!Settings::sent_coords && Settings::requested_coords){
			// 	int* sorted_green_positions = (int*)malloc(sizeof(int)*Settings::get_area());
			// 	int* sorted_red_positions = (int*)malloc(sizeof(int)*Settings::get_area());
			// 	mtx.lock();
			// 	hipMemcpy(temporary_green_positions, maximaGreen, sizeof(int)*Settings::get_area(), hipMemcpyDeviceToDevice);
			// 	hipMemcpy(temporary_red_positions, maximaRed, sizeof(int)*Settings::get_area(), hipMemcpyDeviceToDevice);
			// 	mtx.unlock();

			// 	int* count = (int*)malloc(sizeof(int)*2);

			// 	processPoints(temporary_green_positions, temporary_red_positions, sorted_green_positions, sorted_red_positions, count);

			// 	buffer = (char*)malloc(sizeof(int)*(2+count[0]+count[1]));

			// 	if(opt_debug)
			// 		printf("DEBUG: Count Green : %d ; Count Red : %d\n", count[0], count[1]);

			// 	memcpy(&buffer[0], &count[0], sizeof(int));
			// 	memcpy(&buffer[4], sorted_green_positions, count[0]*sizeof(int));
			// 	memcpy(&buffer[4*(1+count[0])], &count[1], sizeof(int));
			// 	memcpy(&buffer[4*(2+count[0])], sorted_red_positions, count[1]*sizeof(int));

			// 	send(client, buffer, sizeof(int)*(2+count[0]+count[1]), 0);

			// 	printf("INFO: Sent the found locations\n");

			// 	free(buffer);
			// 	free(count);
			// 	free(sorted_green_positions);
			// 	free(sorted_red_positions);

			// 	Settings::set_sent_coords(true);
			// 	Settings::set_requested_coords(false);
			// }

			if (Settings::force_exit) break;
		}
	}

	printf("INFO: datasend_thread: ended\n");
}

void imgproc_thread(){
	printf("INFO: imgproc_thread: started\n");
	
	// Initialize the BackPropagator for the green image
	BackPropagator backprop_G(Settings::values[STG_WIDTH], Settings::values[STG_HEIGHT], LAMBDA_GREEN, (float)Settings::values[STG_Z_GREEN]/(float)1000000);

	// Initialize the BeadFinder
	BeadsFinder beadsFinder(Settings::values[STG_WIDTH], Settings::values[STG_HEIGHT], (uint8_t)Settings::values[STG_IMGTHRS]);

	// Allocate the memory for the images
	G.create(Settings::values[STG_WIDTH], Settings::values[STG_HEIGHT]);
	R.create(Settings::values[STG_WIDTH], Settings::values[STG_HEIGHT]);
	G_backprop.create(Settings::values[STG_WIDTH], Settings::values[STG_HEIGHT]);

	// Allocate the memory for the backprop image on the host
	// uint8_t* hG_backprop = (uint8_t*)malloc(sizeof(uint8_t)*Settings::get_area());

	while(!Settings::force_exit) {
		auto t_cycle_start = std::chrono::system_clock::now();

		// wait tiil new image is ready
		while(Camera::img_produced == Camera::img_processed && !Settings::force_exit) {
			usleep(500);
		}

		// Make copies of red and green channel
		auto t_cp_start = std::chrono::system_clock::now();
		Camera::G.copyTo(G);
		Camera::R.copyTo(R);
		auto t_cp_end = std::chrono::system_clock::now();

		// increase the number of processed images so that the camera starts capturing a new image
		++Camera::img_processed;

		// process the image
		// backprop
		auto t_backprop_start = std::chrono::system_clock::now();
		backprop_G.backprop(G, G_backprop);
		auto t_backprop_end = std::chrono::system_clock::now();

		// Update the image in beadsFinder where the beads are to be searched for
		auto t_beadsfinder_cp_start = std::chrono::system_clock::now();
		beadsFinder.updateImage(G_backprop);
		auto t_beadsfinder_cp_end = std::chrono::system_clock::now();

		// find the bads
		auto t_beadsfinder_start = std::chrono::system_clock::now();
		beadsFinder.findBeads();
		bead_count = beadsFinder.copyPositionsTo(bead_positions);
		auto t_beadsfinder_end = std::chrono::system_clock::now();


		auto t_cycle_end = std::chrono::system_clock::now();
		if(Options::verbose) {
			chrono::duration<double> cycle_elapsed_seconds = t_cycle_end - t_cycle_start;
			chrono::duration<double> cp_elapsed_seconds = t_cp_end - t_cp_start;
			chrono::duration<double> bp_elapsed_seconds = t_backprop_end - t_backprop_start;
			chrono::duration<double> bf_cp_elapsed_seconds = t_beadsfinder_cp_end - t_beadsfinder_cp_start;
			chrono::duration<double> bf_elapsed_seconds = t_beadsfinder_end - t_beadsfinder_start;

			std::cout << "TRACE: Backprop: " << bp_elapsed_seconds.count();
			std::cout << "| BF.cp: " << bf_cp_elapsed_seconds.count();
			std::cout << "| BF.findBeads: " << bf_elapsed_seconds.count();
			std::cout << "| cp: " << cp_elapsed_seconds.count();
			std::cout << "| whole cycle: " << cycle_elapsed_seconds.count();
			std::cout << "| #points: " << bead_count << " s\n";
		}
	}
	
	printf("INFO: imgproc_thread: ended\n");
}

void display_thread(){
	printf("INFO: display_thread: started\n");
	
	char ret_key;
	char filename [50];

	while(true){
		while(Settings::sleeping && Settings::connected && !Settings::force_exit){}
		if (Settings::force_exit) break;

		// Allocate the memory
		ImageData<uint8_t> G_backprop_copy(Settings::values[STG_WIDTH], Settings::values[STG_HEIGHT]);
		ImageData<uint8_t> G_copy(Settings::values[STG_WIDTH], Settings::values[STG_HEIGHT]);
		ImageData<uint8_t> R_copy(Settings::values[STG_WIDTH], Settings::values[STG_HEIGHT]);

		if (Options::show) {
			cv::namedWindow("Basic Visualization", cv::WINDOW_NORMAL);
			cv::setWindowProperty("Basic Visualization", cv::WND_PROP_FULLSCREEN, cv::WINDOW_FULLSCREEN);
			//set the callback function for any mouse event
			if (Options::mousekill) {
				cv::setMouseCallback("Basic Visualization", mouseEventCallback, NULL);
			}
		}

		while(!Settings::initialized && Settings::connected && !Settings::force_exit){}
		if (Settings::force_exit){
			break;
		} 

		const cv::cuda::GpuMat c_img_resized(cv::Size(800, 800), CV_8U);
		const cv::cuda::GpuMat c_img_flip(cv::Size(800, 800), CV_8U);
		const cv::Mat img_disp(cv::Size(800, 800), CV_8U);

		uint32_t last_img_processed = Camera::img_processed;

		while(!Settings::sleeping && Settings::connected){
			if(Camera::img_processed - last_img_processed > 3){
				auto start = std::chrono::system_clock::now();
				if (Options::show && !Options::saveimgs)
					G_backprop.copyTo(G_backprop_copy);
				if (Options::saveimgs) {
					G_backprop.copyTo(G_backprop_copy);
					G.copyTo(G_copy);
					R.copyTo(R_copy);
				}
				
				if (Options::saveimgs) {					
					const cv::Mat G_img(cv::Size(Settings::values[STG_WIDTH], Settings::values[STG_HEIGHT]), CV_8U, G_copy.hostPtr());
					const cv::Mat R_img(cv::Size(Settings::values[STG_WIDTH], Settings::values[STG_HEIGHT]), CV_8U, R_copy.hostPtr());
					const cv::Mat G_backprop_img(cv::Size(Settings::values[STG_WIDTH], Settings::values[STG_HEIGHT]), CV_8U, G_backprop_copy.hostPtr());
					
					sprintf (filename, "./imgs/G_%05d.png", img_count);
					cv::imwrite( filename, G_img );
					
					sprintf (filename, "./imgs/R_%05d.png", img_count);
					cv::imwrite( filename, R_img );
					
					sprintf (filename, "./imgs/G_bp_%05d.png", img_count);
					cv::imwrite( filename, G_backprop_img );
				}				
				
				if (Options::show) {
					const cv::cuda::GpuMat c_img(cv::Size(Settings::values[STG_WIDTH], Settings::values[STG_HEIGHT]), CV_8U, G_backprop_copy.devicePtr());

					// Resize the image so that it fits the display
					cv::cuda::resize(c_img, c_img_resized, cv::Size(800, 800));	
					
					// Flip the axis so that the displayed image corresponds to the actual top view on the image sensor
					cv::cuda::flip(c_img_resized, c_img_flip, 0);

					c_img_flip.download(img_disp);

					// Draw bead positions
					for(int i = 0; i < bead_count; i++) {
						uint32_t x = (bead_positions[2*i]*800)/Settings::values[STG_WIDTH];
						uint32_t y = 800 - (bead_positions[2*i+1]*800)/Settings::values[STG_HEIGHT] - 1;
						cv::circle(img_disp, cv::Point(x, y), 20, 255);
					}
					
					cv::imshow("Basic Visualization", img_disp);
					auto end = std::chrono::system_clock::now();
					std::chrono::duration<double> elapsed_seconds = end-start;
					if(Options::verbose) {
						std::cout << "TRACE: Stroring the image took: " << elapsed_seconds.count() << "s\n";
					}
	
					ret_key = (char) cv::waitKey(1);
					if (ret_key == 27 || ret_key == 'x') Settings::set_force_exit(true);  // exit the app if `esc' or 'x' key was pressed.					
				}				

				img_count++;
				last_img_processed = Camera::img_processed;
			}
			else{
				usleep(1000);
			}

			if (Settings::force_exit) break;
		}
	}

	printf("INFO: display_thread: ended\n");
}


int main(int argc, char* argv[]){
	Options::parse(argc, argv);
	
	if(Options::debug){
		printf("DEBUG: Initial settings:");
		Settings::print();
	}

	if (Options::show) {
		Settings::set_initialized(true);
		Settings::set_connected(true);
		Settings::set_sleeping(false);
	}

	thread camera_thr (Camera::camera_thread);
	thread imgproc_thr (imgproc_thread);
	thread display_thr (display_thread);
	thread network_thr (network_thread);
	thread datasend_thr (datasend_thread);
	thread keyboard_thr (keyboard_thread);
	
	camera_thr.join();
	imgproc_thr.join();
	display_thr.join();
	datasend_thr.join();

	return 0;
}
