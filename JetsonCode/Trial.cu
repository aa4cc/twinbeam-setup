#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "cudaEGL.h"
#include "cuda_egl_interop.h"
#include "thrust/copy.h"
#include "thrust/execution_policy.h"
#include "thrust/device_ptr.h"
#include "Argus/Argus.h"
#include "EGLStream/EGLStream.h"
#include "stdio.h"
#include "stdlib.h"
#include "EGL/egl.h"
#include <iterator>
#include <unistd.h>
#include <iostream>
#include <fstream>
#include <chrono>
#include <opencv2/opencv.hpp>
#include <opencv2/highgui/highgui.hpp>
#include "Kernels.h"
#include <cstdlib>
#include <thread>
#include <mutex>
#include <sys/socket.h>
#include <netinet/in.h>
#include <arpa/inet.h>
#include <string.h>
#include "cxxopts.hpp"
#include "Definitions.h"
#include "Misc.h"
#include "Settings.h"

#define STG_WIDTH Settings::values[STG_WIDTH]
#define STG_HEIGHT Settings::values[STG_HEIGHT]

static const int    DEFAULT_FPS        = 30;

using namespace std;
using namespace Argus;
using namespace EGLStream;

hipError_t res;

float* doubleArray;
float* outputArray;
float* convoOutputArray;
float* convoOutputArrayRed;
hipfftComplex* kernelGreen;
hipfftComplex* kernelRed;

float* redConverted;

float* convolutionMaskGreen;
float* convolutionMaskRed;
float* convoOutputArrayGreen;

float* maximaRed;
float* maximaGreen;
float* doubleTemporary;

int* greenPoints;
int* redPoints;
int* positionsGreen;
int* positionsRed;

int* redPointsLast;
int* greenPointsLast;
int* current_index;

hipfftComplex* convolutionFilterBlur;

int client;

// Options
bool opt_verbose	= false;
bool opt_debug		= false;
bool opt_show		= false;

uint16_t *R;
uint16_t *G;

mutex mtx;

int numBlocks;
short cycles;
std::chrono::duration<double> elapsed_seconds_average;

EGLStreamKHR eglStream;
const textureReference* uvTex;
const textureReference* yTex;

texture<unsigned char, 2, hipReadModeElementType> yTexRef;
texture<uchar2, 2, hipReadModeElementType> uvTexRef;

struct is_not_zero
{
	__host__ __device__
	bool operator()(const int x)
	{
		return x != 0;
	}
};

// cxxopts.hpp related definitions
cxxopts::ParseResult
parse(int argc, char* argv[])
{
  try
  {
    cxxopts::Options options(argv[0], " - Twin-beam setup - image processing");
    options
      .positional_help("[optional args]")
      .show_positional_help();

    options
      .add_options()
      ("s,show", "Display the processed image on the display", 	cxxopts::value<bool>(opt_show))
      ("d,debug", "Prints debug information",					cxxopts::value<bool>(opt_debug))
      ("v,verbose", "Prints some additional information",		cxxopts::value<bool>(opt_verbose))
      ("help", "Prints help")
    ;

    auto result = options.parse(argc, argv);

    if (result.count("help"))
    {
      std::cout << options.help({"", "Group"}) << std::endl;
      exit(0);
    }


    if (opt_debug) {
	    if (opt_show)
	    {
	      std::cout << "Saw option ‘s’" << std::endl;
	    }

	    if (opt_debug)
	    {
	      std::cout << "Saw option ‘d’" << std::endl;
	    }

	    if (opt_verbose)
	    {
	      std::cout << "Saw option ‘v’" << std::endl;
	    }
	}


    return result;

  } catch (const cxxopts::OptionException& e)
  {
    std::cout << "error parsing options: " << e.what() << std::endl;
    exit(1);
  }
}

void processPoints(float* greenInputPoints, float* redInputPoints, int* outputGreenCoords, int* outputRedCoords, int* h_count){
	float* points;
	int* greenCoords;
	int* sortedGreenCoords;
	int* redCoords;
	int* sortedRedCoords;
	
	hipMalloc(&points, 2*Settings::get_area()*sizeof(float));
	hipMalloc(&greenCoords, Settings::get_area()*sizeof(int));
	hipMalloc(&redCoords, Settings::get_area()*sizeof(int));
	hipMalloc(&sortedGreenCoords, Settings::get_area()*sizeof(int));
	hipMalloc(&sortedRedCoords, Settings::get_area()*sizeof(int));

	hipMemcpy(points, greenInputPoints, sizeof(float)*Settings::get_area(), hipMemcpyDeviceToDevice);
	hipMemcpy(&points[Settings::get_area()], redInputPoints, sizeof(float)*Settings::get_area(), hipMemcpyDeviceToDevice);
	findPoints<<<numBlocks, BLOCKSIZE>>>(STG_WIDTH, STG_HEIGHT, points, greenCoords);
	findPoints<<<numBlocks, BLOCKSIZE>>>(STG_WIDTH, STG_HEIGHT, &points[Settings::get_area()], redCoords);
	thrust::device_ptr<int> greenCoordsPtr(greenCoords);
	thrust::device_ptr<int> redCoordsPtr(redCoords);

	thrust::device_ptr<int> sortedGreenCoordsPtr(sortedGreenCoords);
	thrust::device_ptr<int> sortedRedCoordsPtr(sortedRedCoords);

	auto endGreenPointer = thrust::copy_if(thrust::device, greenCoordsPtr, greenCoordsPtr+Settings::get_area(), sortedGreenCoordsPtr, is_not_zero());
	auto endRedPointer = thrust::copy_if(thrust::device, redCoordsPtr, redCoordsPtr+Settings::get_area(), sortedRedCoordsPtr, is_not_zero());

	h_count[0] = (int)(endGreenPointer - sortedGreenCoordsPtr);
	h_count[1] = (int)(endRedPointer - sortedRedCoordsPtr);

	hipMemcpy(outputGreenCoords, sortedGreenCoords, sizeof(int)*h_count[0], hipMemcpyDeviceToHost);
	hipMemcpy(outputRedCoords, sortedRedCoords, sizeof(int)*h_count[1], hipMemcpyDeviceToHost);

	hipFree(points);
	hipFree(greenCoords);
	hipFree(redCoords);
	hipFree(sortedRedCoords);
	hipFree(sortedGreenCoords);
}

//#region

__global__ void yuv2bgr(int width, int height, int offset_x, int offset_y,
						uint16_t* G, uint16_t* R)
        {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            int count = width*height;
            int tx, ty, ty2;
            float y1, y2;
            float u1, v2, v1;
            for (int i = index; i < count; i += stride)
            {
            	ty = i/width + offset_y;
            	ty2 = i/width + offset_y - (512-50);
            	tx = i%width + offset_x;
            	y1 = (float)((tex2D<unsigned char>(yTexRef, (float)tx+0.5f, (float)ty+0.5f) - (float)16) * 1.164383f);
            	y2 = (float)((tex2D<unsigned char>(yTexRef, (float)tx+0.5f, (float)ty2+0.5f) - (float)16) * 1.164383f);
            	u1 = (float)(tex2D<uchar2>(uvTexRef, (float)(tx/2)+(float)(tx%2)+0.5f,
            	 	 (float)(ty/2)+(float)(ty%2)+0.5f).x - 128) * 0.391762f;
            	v2 = (float)(tex2D<uchar2>(uvTexRef, (float)(tx/2)+(float)(tx%2)+0.5f,
            	     (float)(ty2/2)+(float)(ty2%2)+0.5f).y - 128) * 1.596027f;
            	v1 = (float)(tex2D<uchar2>(uvTexRef, (float)(tx/2)+(float)(tx%2)+0.5f,
            	     (float)(ty/2)+(float)(ty%2)+0.5f).y - 128) * 0.812968f;
				G[i] = (uint16_t)(y1-u1-v1);
				R[i] = (uint16_t)(y2+v2+u1/10);
            }
        }

void transformKernel(int M, int N, int kernelDim, float* kernel, hipfftComplex* outputKernel){
	kernelToImage<<<numBlocks, BLOCKSIZE>>>(M, N, kernelDim, kernel, outputKernel);
    hipfftHandle plan;
    hipfftPlan2d(&plan, N,M, HIPFFT_C2C);
    hipfftExecC2C(plan, outputKernel, outputKernel, HIPFFT_FORWARD);
    hipfftDestroy(plan);
}


void h_backPropagate(int M, int N, float lambda, float z, float* input,
		hipfftComplex* kernel, float* output, float* output2, bool display)
{
    hipfftComplex* doubleComplexArray;
    hipfftComplex* Hq;
    hipfftComplex* image;
    hipfftComplex* kernelizedImage;
    float* temporary;
    float* extremes;
    hipfftHandle plan;

    hipMalloc(&doubleComplexArray, 3*N*M*sizeof(hipfftComplex));
    Hq = &doubleComplexArray[0];
    image = &doubleComplexArray[N*M];
    kernelizedImage = &doubleComplexArray[2*N*M];
    hipMalloc(&temporary, N*M*sizeof(float));
    hipMalloc(&extremes, sizeof(float));

    convertToComplex<<<numBlocks, BLOCKSIZE>>>(N*M, input, image);
    // Declaring the FFT plan
    hipfftPlan2d(&plan, N,M, HIPFFT_C2C);
    // Execute forward FFT on the green channel
    hipfftExecC2C(plan, image, image, HIPFFT_FORWARD);
    // Calculating the Hq matrix according to the equations in the original .m file.
    calculate<<<numBlocks, BLOCKSIZE>>>(N,M, z, PIXEL_DX, REFRACTION_INDEX, lambda, Hq);
    // Element-wise multiplication of Hq matrix and the image
	elMultiplication<<<numBlocks, BLOCKSIZE>>>(M, N, Hq, image);
	blurFilter<<<numBlocks, BLOCKSIZE>>>(M, N, 3, image);
	elMultiplication2<<<numBlocks, BLOCKSIZE>>>(M, N, image, kernel, kernelizedImage);
    if(display){
		// Executing inverse FFT
		hipfftExecC2C(plan, image, image, HIPFFT_BACKWARD);
		// Conversion of result matrix to a real double matrix
		absoluteValue<<<numBlocks, BLOCKSIZE>>>(M,N, image, output);

		findExtremes<<<numBlocks, BLOCKSIZE>>>(M,N, output, extremes);
		normalize<<<numBlocks, BLOCKSIZE>>>(M,N, output, extremes);
	}
	hipfftExecC2C(plan, kernelizedImage, kernelizedImage, HIPFFT_BACKWARD);
	cutAndConvert<<<numBlocks, BLOCKSIZE>>>(M,N,kernelizedImage, convoOutputArrayGreen);
    hipFree(extremes);
    hipMalloc(&extremes, sizeof(float));
	findExtremes<<<numBlocks, BLOCKSIZE>>>(M,N, convoOutputArrayGreen, extremes);
	normalize<<<numBlocks, BLOCKSIZE>>>(M,N, convoOutputArrayGreen, extremes);
	getLocalMaxima<<<numBlocks, BLOCKSIZE>>>(M,N,convoOutputArrayGreen,output2);
	// Freeing the memory of FFT plan
	hipfftDestroy(plan);

    hipFree(extremes);
    hipFree(doubleComplexArray);
    hipFree(temporary);
}

void keyboard_thread(){
	printf("keyboard_thread: started\n");

	char input;
	while(!Settings::force_exit){
		input = getchar();
		if(input == 's'){
			printf("Putting the process to sleep\n");
			Settings::set_sleeping(true);
			Settings::set_initialized(false);
		}
		else if(input == 'c'){
			printf("Connected the main manipulation computer\n");
			Settings::set_connected(true);
		}
		else if(input == 'w'){
			printf("Starting the program from keyboard\n");
			Settings::set_initialized(true);
			Settings::set_sleeping(false);
		}
		else if(input == 'd'){
			Settings::set_connected(false);
			Settings::set_sleeping(true);
			Settings::set_initialized(false);
		}
		else if(input == 'e'){
			Settings::set_force_exit(true);
		}		
	}

	printf("keyboard_thread: ended\n");
}

void changeSettings(char* buf){
	int tmpSettings; 
	int count = 0;
	int current_index = 2;
	while(count < 7){
		string str = "";
		while(isdigit(buf[current_index])){
			str.append(1u,buf[current_index]);
			current_index++;
		}
		try{
			tmpSettings = atol(str.c_str());
		}
		catch(int e ){
			printf("Number is too large\n");
			tmpSettings = 0;
		}
		if(tmpSettings != 0){
			Settings::set_setting(count, tmpSettings);
			printf("%d\n", Settings::values[count]);
		}
		count++;
		current_index++;
	}
}

void input_thread(){
	printf("input_thread: started\n");

	std::string text;
	sockaddr_in sockName;
	sockaddr_in clientInfo; 
	int mainSocket;
	char buf[BUFSIZE];
	socklen_t addrlen;
	MESSAGE_TYPE response;
	
	mainSocket = socket(AF_INET, SOCK_STREAM, IPPROTO_TCP);
	if(mainSocket == -1)
		printf("Couldn't create socket!\n");
	sockName.sin_family = AF_INET;
	sockName.sin_port =	htons(PORT);
	sockName.sin_addr.s_addr = INADDR_ANY;

	int yes = 1;
	if ( setsockopt(mainSocket, SOL_SOCKET, SO_REUSEADDR, &yes, sizeof(int)) == -1 )
	{
	    perror("setsockopt");
	}

	bind(mainSocket, (sockaddr*)&sockName, sizeof(sockName));
	listen(mainSocket, 10000000);
	while(!Settings::force_exit){
		
		addrlen = sizeof(clientInfo);
		client = accept(mainSocket, (sockaddr*)&clientInfo, &addrlen);
		cout << "Got a connection from " << inet_ntoa((in_addr)clientInfo.sin_addr) << endl;
		if (client != -1)
		 {
			 Settings::set_connected(true);
		 }

		while(Settings::connected && !Settings::force_exit){
			int msg_len = recv(client, buf, BUFSIZE - 1, 0);

			if (msg_len == -1)
			{
				printf("Error while receiving data\n");
			}

			printf("Received bytes: %d\n", msg_len);

			response = parseMessage(buf);
			switch(response){
				case MSG_WAKEUP:
				{
					Settings::set_sleeping(false);
					Settings::set_initialized(true);
					break;
				}
				case MSG_SLEEP:
				{
					Settings::set_sleeping(true);
					Settings::set_initialized(false);
					break;
				}
				case MSG_SETTINGS:
				{
					if(!Settings::sleeping)
						printf("Can't change settings while the loop is running\n");
					else{
						changeSettings(buf);
						printf("Changed settings\n");
					}
					break;
				}
				case MSG_DISCONNECT:
				{
					Settings::set_connected(false);
					Settings::set_sleeping(true);
					Settings::set_initialized(false);
					break;
				}
				case MSG_REQUEST:
					Settings::set_requested_image(true);
					Settings::set_requested_type(BACKPROPAGATED);
					break;
				case MSG_REQUEST_RAW_G:
					Settings::set_requested_image(true);
					Settings::set_requested_type(RAW_G);
					break;
				case MSG_REQUEST_RAW_R:
					Settings::set_requested_image(true);
					Settings::set_requested_type(RAW_R);
					break;
				case MSG_COORDS:
					Settings::set_requested_coords(true);
					break;
				case MSG_HELLO:
				{
					send(client, "Hello!",7,0);
					break;
				}	
			} 
		}
		close(client);
	}
	close(mainSocket);

	printf("input_thread: ended\n");
}

void consumer_thread(){
	printf("consumer_thread: started\n");
	//Initializing LibArgus according to the tutorial for a sample project.
	// First we create a CameraProvider, necessary for each project.
	UniqueObj<CameraProvider> cameraProvider(CameraProvider::create());
	ICameraProvider* iCameraProvider = interface_cast<ICameraProvider>(cameraProvider);
	if(!iCameraProvider){
		printf("Failed to establish libargus connection\n");
	}
	
	// Second we select a device from which to receive pictures (camera)
	std::vector<CameraDevice*> cameraDevices;
	iCameraProvider->getCameraDevices(&cameraDevices);
	if (cameraDevices.size() == 0){
		printf("No camera devices available\n");
	}
	CameraDevice *selectedDevice = cameraDevices[0];

	// We create a capture session 
	UniqueObj<CaptureSession> captureSession(iCameraProvider->createCaptureSession(selectedDevice));
	ICaptureSession *iCaptureSession = interface_cast<ICaptureSession>(captureSession);
	if (!iCaptureSession){
 		printf("Failed to create CaptureSession\n");
	}
	
	//CUDA variable declarations
	cudaEglStreamConnection conn;
	hipGraphicsResource_t resource;
	cudaEglFrame eglFrame;		
	hipArray_t yArray;
	hipArray_t uvArray;
	hipChannelFormatDesc yChannelDesc;
	hipChannelFormatDesc uvChannelDesc;

	while(!Settings::force_exit){
		while(Settings::connected && !Settings::force_exit){
			while(Settings::sleeping && Settings::connected && !Settings::force_exit){}
			if (Settings::force_exit) break;
			// Managing the settings for the capture session.
			UniqueObj<OutputStreamSettings> streamSettings(iCaptureSession->createOutputStreamSettings());
			IOutputStreamSettings *iStreamSettings = interface_cast<IOutputStreamSettings>(streamSettings);
			iStreamSettings->setPixelFormat(PIXEL_FMT_YCbCr_420_888);
			iStreamSettings->setResolution(Size2D<uint32_t>(WIDTH,HEIGHT));
			
			// Creating an Output stream. This should already create a producer.
			UniqueObj<OutputStream> outputStream(iCaptureSession->createOutputStream(streamSettings.get()));
			IStream* iStream = interface_cast<IStream>(outputStream);
			if (!iStream){
				printf("Failed to create OutputStream\n");
			}
			eglStream = iStream->getEGLStream();
			cudaEGLStreamConsumerConnect(&conn, eglStream);
			
			// Managing requests.
			UniqueObj<Request> request(iCaptureSession->createRequest());
			IRequest *iRequest = interface_cast<IRequest>(request);
			iRequest->enableOutputStream(outputStream.get());
			
			ISourceSettings *iSourceSettings = interface_cast<ISourceSettings>(iRequest->getSourceSettings());
			iSourceSettings->setFrameDurationRange(Range<uint64_t>(1e9/DEFAULT_FPS));
			iSourceSettings->setExposureTimeRange(Range<uint64_t>(Settings::values[STG_EXPOSURE],Settings::values[STG_EXPOSURE]));
			iSourceSettings->setGainRange(Range<float>(0.5,1.5));

			IAutoControlSettings *iAutoSettings = interface_cast<IAutoControlSettings>(iRequest->getAutoControlSettings());
			iAutoSettings->setExposureCompensation(0);
			iAutoSettings->setIspDigitalGainRange(Range<float>(0,0));
			iAutoSettings->setWbGains(100);
			iAutoSettings->setColorSaturation(1.0);
			iAutoSettings->setColorSaturationBias(1.0);
			iAutoSettings->setColorSaturationEnable(true);
			iAutoSettings->setAwbLock(true);
			iAutoSettings->setAeAntibandingMode(AE_ANTIBANDING_MODE_OFF);

			IDenoiseSettings *iDenoiseSettings = interface_cast<IDenoiseSettings>(request);	
			iDenoiseSettings->setDenoiseMode(DENOISE_MODE_FAST);
			iDenoiseSettings->setDenoiseStrength(1.0);

			hipMalloc(&G, Settings::get_area()*sizeof(uint16_t));
			hipMalloc(&R, Settings::get_area()*sizeof(uint16_t));
			hipMalloc(&positionsGreen, Settings::get_area()*sizeof(float));
			hipMalloc(&positionsRed, Settings::get_area()*sizeof(float));		
			
			hipMalloc(&convolutionMaskGreen, CONVO_DIM_GREEN*CONVO_DIM_GREEN*sizeof(float));
			hipMalloc(&convolutionMaskRed, CONVO_DIM_RED*CONVO_DIM_RED*sizeof(float));
			numBlocks = 1024;
			generateConvoMaskGreen<<<numBlocks, BLOCKSIZE>>>(CONVO_DIM_GREEN, CONVO_DIM_GREEN, convolutionMaskGreen);
			generateConvoMaskRed<<<numBlocks, BLOCKSIZE>>>(CONVO_DIM_RED, CONVO_DIM_RED, convolutionMaskRed);
			
			hipMalloc(&kernelGreen, Settings::get_area()*sizeof(hipfftComplex));
			hipMalloc(&kernelRed, Settings::get_area()*sizeof(hipfftComplex));

			hipMalloc(&convolutionFilterBlur, Settings::get_area()*sizeof(hipfftComplex));
			generateBlurFilter<<<numBlocks, BLOCKSIZE>>>(STG_WIDTH, STG_HEIGHT, convolutionFilterBlur);
			
			transformKernel(STG_WIDTH, STG_HEIGHT, CONVO_DIM_GREEN, convolutionMaskGreen, kernelGreen);
			transformKernel(STG_WIDTH, STG_HEIGHT, CONVO_DIM_RED, convolutionMaskRed, kernelRed);
			
			hipMalloc(&convoOutputArrayGreen, Settings::get_area()*sizeof(float));
			hipMalloc(&convoOutputArrayRed, Settings::get_area()*sizeof(float));
			hipMallocManaged(&current_index, sizeof(int));
			mtx.lock();
			hipMalloc(&maximaGreen, Settings::get_area()*sizeof(float));
			hipMalloc(&maximaRed, Settings::get_area()*sizeof(float));
			hipMalloc(&doubleArray, 2*Settings::get_area()*sizeof(float));
			doubleTemporary = &doubleArray[0];
			outputArray = &doubleArray[Settings::get_area()];
			hipMalloc(&convoOutputArray, Settings::get_area()*sizeof(float));
			hipMalloc(&redConverted, Settings::get_area()*sizeof(float));
			mtx.unlock();

			yTexRef.normalized = 0;
			yTexRef.filterMode = hipFilterModePoint;
			yTexRef.addressMode[0] = hipAddressModeClamp;
			yTexRef.addressMode[1] = hipAddressModeClamp;
			hipGetTextureReference(&yTex, HIP_SYMBOL(&yTexRef));
			
			uvTexRef.normalized = 0;
			uvTexRef.filterMode = hipFilterModePoint;
			uvTexRef.addressMode[0] = hipAddressModeClamp;
			uvTexRef.addressMode[1] = hipAddressModeClamp;
			hipGetTextureReference(&uvTex, HIP_SYMBOL(&uvTexRef));
			
			//CUDA initialization
			//Main loop
			auto initializer = std::chrono::system_clock::now();
			std::chrono::duration<double> elapsed_seconds_average = initializer-initializer;

			int final_count = 0;
			while(!Settings::initialized && Settings::connected && !Settings::force_exit){}
			if (Settings::force_exit) break;

			while(!Settings::sleeping && Settings::connected && ! Settings::force_exit){
				auto start = std::chrono::system_clock::now();
				
				
				iCaptureSession->capture(request.get());
				res = cudaEGLStreamConsumerAcquireFrame(&conn, &resource, 0, 5000);
				if(res != hipSuccess){
					continue;
				}
				cudaGraphicsResourceGetMappedEglFrame(&eglFrame, resource, 0, 0);
				yArray = eglFrame.frame.pArray[0];
				uvArray = eglFrame.frame.pArray[1];
				
				hipGetChannelDesc(&yChannelDesc, (hipArray_const_t)(yArray));
				hipBindTextureToArray(yTex, (hipArray_const_t)(yArray), &yChannelDesc);
				hipGetChannelDesc(&uvChannelDesc, (hipArray_const_t)(uvArray));
				hipBindTextureToArray(uvTex, (hipArray_const_t)(uvArray), &uvChannelDesc);
				auto initialization = std::chrono::system_clock::now();

				
				numBlocks = (Settings::get_area()/2 +BLOCKSIZE -1)/BLOCKSIZE;
				yuv2bgr<<<numBlocks, BLOCKSIZE>>>(STG_WIDTH, STG_HEIGHT,
												Settings::values[STG_OFFSET_X], Settings::values[STG_OFFSET_Y], G, R);
				auto test = std::chrono::system_clock::now();
				u16ToDouble<<<numBlocks, BLOCKSIZE>>>(STG_WIDTH, STG_HEIGHT, G, doubleTemporary);
				u16ToDouble<<<numBlocks, BLOCKSIZE>>>(STG_WIDTH, STG_HEIGHT, R, redConverted);
				mtx.lock();
				h_backPropagate(STG_WIDTH, STG_HEIGHT, LAMBDA_GREEN, (float)Settings::values[STG_Z_GREEN]/(float)1000000,
						doubleTemporary, kernelGreen, outputArray, maximaGreen, true);		
				h_backPropagate(STG_WIDTH,STG_HEIGHT, LAMBDA_RED, (float)Settings::values[STG_Z_RED]/(float)1000000,
						redConverted, kernelRed, convoOutputArray, maximaRed, false);
				mtx.unlock();
				
				
				auto test2 = std::chrono::system_clock::now();
				std::chrono::duration<double> elapsed_seconds = test2-test;
				
				if(opt_verbose) {
					std::cout << "Converting the image format took: " << elapsed_seconds.count() << "s\n";
				}

				hipUnbindTexture(yTex);
				hipUnbindTexture(uvTex);
				
				cudaEGLStreamConsumerReleaseFrame(&conn, resource, 0);
				
				auto end = std::chrono::system_clock::now();
				elapsed_seconds = end-start;
				elapsed_seconds_average +=elapsed_seconds;
				final_count++;
				Settings::sent_coords = false;
				
				if(opt_verbose) {
					std::cout << "This cycle took: " << elapsed_seconds.count() << "s\n";
				}

				cycles++;				
			}
			std::cout << "average complete: " << elapsed_seconds_average.count()/final_count << "s\n";
			iCaptureSession->waitForIdle();
			
			hipFree(G);
			hipFree(R);
			
			hipFree(doubleArray);
			hipFree(convoOutputArray);
			hipFree(redConverted);
			hipFree(maximaGreen);
			hipFree(maximaRed);
			hipFree(convolutionMaskGreen);
			hipFree(convolutionMaskRed);
			hipFree(convoOutputArrayRed);
			hipFree(kernelGreen);
			hipFree(kernelRed);
			
			cudaEGLStreamConsumerDisconnect(&conn);
			iStream->disconnect();
			outputStream.reset();
		}
	}

	printf("consumer_thread: ended\n");
}

void CallBackFunc(int event, int x, int y, int flags, void* userdata)
{
	// https://www.opencv-srf.com/2011/11/mouse-events.html
	if ( event == CV_EVENT_MOUSEMOVE )
     {
        cout << "Mouse move over the window - position (" << x << ", " << y << ")" << endl;
        if (Settings::touch_kill) {
        	Settings::set_force_exit(true);
      	}
     }
}

void output_thread(){
	printf("output_thread: started\n");

	float *temporary;
	float *temporary_red_positions;
	float *temporary_green_positions;
	char* buffer;
	while(true){
		while(Settings::sleeping && !Settings::force_exit){}
		if (Settings::force_exit) break;

		hipMalloc(&temporary, Settings::get_area()*sizeof(float));
		hipMalloc(&temporary_red_positions, Settings::get_area()*sizeof(float));
		hipMalloc(&temporary_green_positions, Settings::get_area()*sizeof(float));
		
		while(Settings::connected && !Settings::sleeping){
			if(Settings::requested_image){
				mtx.lock();
				switch (Settings::requested_type){
					case BACKPROPAGATED:
						hipMemcpy(temporary, outputArray, sizeof(float)*Settings::get_area(), hipMemcpyDeviceToDevice);
						break;
					case RAW_G:
						hipMemcpy(temporary, doubleTemporary, sizeof(float)*Settings::get_area(), hipMemcpyDeviceToDevice);
						break;
					case RAW_R:
						hipMemcpy(temporary, redConverted, sizeof(float)*Settings::get_area(), hipMemcpyDeviceToDevice);
						break;
				}	

				mtx.unlock();

				buffer = (char*)malloc(Settings::get_area()*sizeof(float));
				hipMemcpy(buffer, temporary, sizeof(float)*Settings::get_area(), hipMemcpyDeviceToHost);
				send(client, buffer, sizeof(float)*Settings::get_area(), 0);
				free(buffer);
				printf("Image sent!\n");
				Settings::set_requested_image(false);
			}
			if(!Settings::sent_coords && Settings::requested_coords){
				int* sorted_green_positions = (int*)malloc(sizeof(int)*Settings::get_area());
				int* sorted_red_positions = (int*)malloc(sizeof(int)*Settings::get_area());
				mtx.lock();
				hipMemcpy(temporary_green_positions, maximaGreen, sizeof(int)*Settings::get_area(), hipMemcpyDeviceToDevice);
				hipMemcpy(temporary_red_positions, maximaRed, sizeof(int)*Settings::get_area(), hipMemcpyDeviceToDevice);
				mtx.unlock();

				int* count = (int*)malloc(sizeof(int)*2);

				processPoints(temporary_green_positions, temporary_red_positions, sorted_green_positions, sorted_red_positions, count);

				buffer = (char*)malloc(sizeof(int)*(2+count[0]+count[1]));

				memcpy(&buffer[0], &count[0], sizeof(int));
				memcpy(&buffer[4], sorted_green_positions, count[0]*sizeof(int));
				memcpy(&buffer[4*(1+count[0])], &count[1], sizeof(int));
				memcpy(&buffer[4*(2+count[0])], sorted_red_positions, count[1]*sizeof(int));

				send(client, buffer, sizeof(int)*(2+count[0]+count[1]), 0);

				free(buffer);
				free(count);
				free(sorted_green_positions);
				free(sorted_red_positions);

				Settings::set_sent_coords(true);
				Settings::set_requested_coords(false);
			}

			if (Settings::force_exit) break;
		}
		hipFree(temporary_red_positions);
		hipFree(temporary_green_positions);
		
		hipFree(temporary);
	}

	printf("output_thread: ended\n");
}

void print_thread(){
	printf("print_thread: started\n");

	float* tempArray;
	float* tempArray2;
	while(true){
		while(Settings::sleeping && Settings::connected && !Settings::force_exit){}
		if (Settings::force_exit) break;
			
		hipMalloc(&tempArray, sizeof(float)*Settings::get_area());
		hipMalloc(&tempArray2, sizeof(float)*Settings::get_area());
		float* output = (float*)malloc(sizeof(float)*Settings::get_area());
		float* output2 = (float*)malloc(sizeof(float)*Settings::get_area());
		cv::namedWindow("Basic Visualization", CV_WINDOW_NORMAL);
		cv::setWindowProperty("Basic Visualization", CV_WND_PROP_FULLSCREEN, CV_WINDOW_FULLSCREEN);
		//set the callback function for any mouse event
     	cv::setMouseCallback("Basic Visualization", CallBackFunc, NULL);

		while(!Settings::initialized && Settings::connected && !Settings::force_exit){}
		if (Settings::force_exit){
			hipFree(tempArray);
			hipFree(tempArray2);
			free(output);
			free(output2);
			break;
		} 

		while(!Settings::sleeping && Settings::connected){
			if(cycles >= 3){
				cycles = 0;
				mtx.lock();
				hipMemcpy(tempArray, maximaGreen, sizeof(float)*Settings::get_area(), hipMemcpyDeviceToDevice);
				hipMemcpy(tempArray2, outputArray, sizeof(float)*Settings::get_area(), hipMemcpyDeviceToDevice);
				mtx.unlock();

				hipMemcpy(output, tempArray, sizeof(float)*Settings::get_area(), hipMemcpyDeviceToHost);
				hipMemcpy(output2, tempArray2, sizeof(float)*Settings::get_area(), hipMemcpyDeviceToHost);
				printf("%g \n", output2[STG_HEIGHT*STG_HEIGHT/2 + STG_WIDTH]);
				const cv::Mat img(cv::Size(STG_WIDTH, STG_HEIGHT), CV_32F, output);
				const cv::Mat img2(cv::Size(STG_WIDTH, STG_HEIGHT), CV_32F, output2);

				const cv::Mat img2_trans(cv::Size(STG_WIDTH, STG_HEIGHT), CV_32F);
				const cv::Mat img_trans(cv::Size(STG_WIDTH, STG_HEIGHT), CV_32F);

				cv::flip(img2, img2_trans, -1);
				cv::transpose(img2_trans, img2);
				cv::flip(img, img_trans, -1);
				cv::transpose(img_trans, img);

				const cv::Mat result = img2;
				cv::imshow("Basic Visualization", result);
				cv::waitKey(1);
			}
			else{
				usleep(5000);
			}

			if (Settings::force_exit) break;
		}
		hipFree(tempArray);
		hipFree(tempArray2);
		free(output);
		free(output2);
	}

	printf("print_thread: ended\n");
}


int main(int argc, char* argv[]){
	for(int i = 0 ; i < STG_NUMBER_OF_SETTINGS; i++){
		printf("%d\n", Settings::values[i]);
	}
  	auto result = parse(argc, argv);
	
	cycles = 0;

	if (opt_show) {
		Settings::set_initialized(true);
		Settings::set_connected(true);
		Settings::set_sleeping(false);
		Settings::set_touch_kill(true);
	}

	thread consumr_thr (consumer_thread);
	thread print_thr (print_thread);
	thread input_thr (input_thread);
	thread output_thr (output_thread);
	thread keyboard_thr (keyboard_thread);
	
	consumr_thr.join();
	print_thr.join();
	output_thr.join();

	return 0;
}
