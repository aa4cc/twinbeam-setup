#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "cudaEGL.h"
#include "cuda_egl_interop.h"
#include "thrust/copy.h"
#include "thrust/execution_policy.h"
#include "thrust/device_ptr.h"
#include "Argus/Argus.h"
#include "EGLStream/EGLStream.h"
#include "stdio.h"
#include "stdlib.h"
#include "EGL/egl.h"
#include <iterator>
#include <unistd.h>
#include <iostream>
#include <fstream>
#include <chrono>
#include <opencv2/opencv.hpp>
#include <opencv2/highgui/highgui.hpp>
#include "Kernels.h"
#include <cstdlib>
#include <thread>
#include <mutex>
#include <sys/socket.h>
#include <netinet/in.h>
#include <arpa/inet.h>
#include <string.h>
#include "cxxopts.hpp"
#include "Definitions.h"
#include "Misc.h"
#include "Settings.h"
#include "BackPropagator.h"

#define dSTG_WIDTH Settings::values[STG_WIDTH]
#define dSTG_HEIGHT Settings::values[STG_HEIGHT]

static const int    DEFAULT_FPS        = 30;

using namespace std;
using namespace Argus;
using namespace EGLStream;

hipError_t res;

float* redOutputArray;
hipfftComplex* kernelGreen;
hipfftComplex* kernelRed;

float* convolutionMaskGreen;
float* convolutionMaskRed;

float* maximaRed;
float* maximaGreen;

int* greenPoints;
int* redPoints;
int* positionsGreen;
int* positionsRed;

int* redPointsLast;
int* greenPointsLast;

hipfftComplex* convolutionFilterBlur;

int client;

// Options
bool opt_verbose	= false;
bool opt_debug		= false;
bool opt_show		= false;
bool opt_saveimgs	= false;
bool opt_mousekill 	= false;

uint16_t *R;
uint16_t *G;
uint16_t *G_backprop;
float *G_float;

mutex mtx;

int numBlocks;
short cycles;
int final_count;
std::chrono::duration<double> elapsed_seconds_average;

EGLStreamKHR eglStream;
const textureReference* uvTex;
const textureReference* yTex;

texture<unsigned char, 2, hipReadModeElementType> yTexRef;
texture<uchar2, 2, hipReadModeElementType> uvTexRef;

struct is_not_zero
{
	__host__ __device__
	bool operator()(const int x)
	{
		return x != 0;
	}
};

// cxxopts.hpp related definitions
cxxopts::ParseResult
parse(int argc, char* argv[])
{
  try
  {
    cxxopts::Options options(argv[0], " - Twin-beam setup - image processing");
    options
      .positional_help("[optional args]")
      .show_positional_help();

    options
      .add_options()
      ("s,show", "Display the processed image on the display", 	cxxopts::value<bool>(opt_show))
      ("saveimgs", "Save images", 	cxxopts::value<bool>(opt_saveimgs))
      ("d,debug", "Prints debug information",					cxxopts::value<bool>(opt_debug))
      ("k,mousekill", "Moving the mouse or toching the screen kills the app",					cxxopts::value<bool>(opt_mousekill))
      ("v,verbose", "Prints some additional information",		cxxopts::value<bool>(opt_verbose))
      ("help", "Prints help")
    ;
	
    auto result = options.parse(argc, argv);

    if (result.count("help"))
    {
      std::cout << options.help({"", "Group"}) << std::endl;
      exit(0);
    }


    if (opt_debug) {
	    if (opt_show)
	    {
	      std::cout << "Saw option ‘s’" << std::endl;
	    }

	    if (opt_debug)
	    {
	      std::cout << "Saw option ‘d’" << std::endl;
	    }

	    if (opt_verbose)
	    {
	      std::cout << "Saw option ‘v’" << std::endl;
	    }
	}


    return result;

  } catch (const cxxopts::OptionException& e)
  {
    std::cout << "error parsing options: " << e.what() << std::endl;
    exit(1);
  }
}

void processPoints(float* greenInputPoints, float* redInputPoints, int* outputGreenCoords, int* outputRedCoords, int* h_count){
	float* points;
	int* greenCoords;
	int* sortedGreenCoords;
	int* redCoords;
	int* sortedRedCoords;
	
	hipMalloc(&points, 2*Settings::get_area()*sizeof(float));
	hipMalloc(&greenCoords, Settings::get_area()*sizeof(int));
	hipMalloc(&redCoords, Settings::get_area()*sizeof(int));
	hipMalloc(&sortedGreenCoords, Settings::get_area()*sizeof(int));
	hipMalloc(&sortedRedCoords, Settings::get_area()*sizeof(int));

	hipMemcpy(points, greenInputPoints, sizeof(float)*Settings::get_area(), hipMemcpyDeviceToDevice);
	hipMemcpy(&points[Settings::get_area()], redInputPoints, sizeof(float)*Settings::get_area(), hipMemcpyDeviceToDevice);
	findPoints<<<numBlocks, BLOCKSIZE>>>(dSTG_WIDTH, dSTG_HEIGHT, points, greenCoords);
	findPoints<<<numBlocks, BLOCKSIZE>>>(dSTG_WIDTH, dSTG_HEIGHT, &points[Settings::get_area()], redCoords);
	thrust::device_ptr<int> greenCoordsPtr(greenCoords);
	thrust::device_ptr<int> redCoordsPtr(redCoords);

	thrust::device_ptr<int> sortedGreenCoordsPtr(sortedGreenCoords);
	thrust::device_ptr<int> sortedRedCoordsPtr(sortedRedCoords);

	auto endGreenPointer = thrust::copy_if(thrust::device, greenCoordsPtr, greenCoordsPtr+Settings::get_area(), sortedGreenCoordsPtr, is_not_zero());
	auto endRedPointer = thrust::copy_if(thrust::device, redCoordsPtr, redCoordsPtr+Settings::get_area(), sortedRedCoordsPtr, is_not_zero());

	h_count[0] = (int)(endGreenPointer - sortedGreenCoordsPtr);
	h_count[1] = (int)(endRedPointer - sortedRedCoordsPtr);

	hipMemcpy(outputGreenCoords, sortedGreenCoords, sizeof(int)*h_count[0], hipMemcpyDeviceToHost);
	hipMemcpy(outputRedCoords, sortedRedCoords, sizeof(int)*h_count[1], hipMemcpyDeviceToHost);

	hipFree(points);
	hipFree(greenCoords);
	hipFree(redCoords);
	hipFree(sortedRedCoords);
	hipFree(sortedGreenCoords);
}

//#region

__global__ void yuv2bgr(int width, int height, int offset_x, int offset_y,
						uint16_t* G, uint16_t* R)
        {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            int count = width*height;
            int tx, ty, ty2;
            float y1, y2;
            float u1, v2, v1;
            for (int i = index; i < count; i += stride)
            {
            	ty = i/width + offset_y;
            	ty2 = i/width + offset_y - (512);
            	tx = i%width + offset_x;
            	y1 = (float)((tex2D<unsigned char>(yTexRef, (float)tx+0.5f, (float)ty+0.5f) - (float)16) * 1.164383f);
            	y2 = (float)((tex2D<unsigned char>(yTexRef, (float)tx+0.5f, (float)ty2+0.5f) - (float)16) * 1.164383f);
            	u1 = (float)(tex2D<uchar2>(uvTexRef, (float)(tx/2)+(float)(tx%2)+0.5f,
            	 	 (float)(ty/2)+(float)(ty%2)+0.5f).x - 128) * 0.391762f;
            	v2 = (float)(tex2D<uchar2>(uvTexRef, (float)(tx/2)+(float)(tx%2)+0.5f,
            	     (float)(ty2/2)+(float)(ty2%2)+0.5f).y - 128) * 1.596027f;
            	v1 = (float)(tex2D<uchar2>(uvTexRef, (float)(tx/2)+(float)(tx%2)+0.5f,
            	     (float)(ty/2)+(float)(ty%2)+0.5f).y - 128) * 0.812968f;
				G[i] = (uint16_t)(y1-u1-v1);
				R[i] = (uint16_t)(y2+v2+u1/10);
            }
        }

void transformKernel(int M, int N, int kernelDim, float* kernel, hipfftComplex* outputKernel){
	kernelToImage<<<numBlocks, BLOCKSIZE>>>(M, N, kernelDim, kernel, outputKernel);
    hipfftHandle plan;
    hipfftPlan2d(&plan, N,M, HIPFFT_C2C);
    hipfftExecC2C(plan, outputKernel, outputKernel, HIPFFT_FORWARD);
    hipfftDestroy(plan);
}

void keyboard_thread(){
	printf("INFO: keyboard_thread: started\n");

	char input;
	while(!Settings::force_exit){
		input = getchar();
		if(input == 's'){
			printf("INFO: Putting the process to sleep.\n");
			Settings::set_sleeping(true);
			Settings::set_initialized(false);
		}
		else if(input == 'c'){
			printf("INFO: Simulating connection to main computation unit.\n");
			Settings::set_connected(true);
		}
		else if(input == 'w'){
			printf("INFO: Starting the program from keyboard.\n");
			Settings::set_initialized(true);
			Settings::set_sleeping(false);
		}
		else if(input == 'd'){
			Settings::set_connected(false);
			Settings::set_sleeping(true);
			Settings::set_initialized(false);
		}
		else if(input == 'e'){
			Settings::set_force_exit(true);
		}		
	}

	printf("INFO: keyboard_thread: ended\n");
}

void changeSettings(char* buf){
	int tmpSettings; 
	int count = 0;
	int current_index = 2;
	while(count < 7){
		string str = "";
		while(isdigit(buf[current_index])){
			str.append(1u,buf[current_index]);
			current_index++;
		}
		try{
			tmpSettings = atol(str.c_str());
		}
		catch(int e ){
			printf("Number is too large\n");
			tmpSettings = 0;
		}
		if(tmpSettings != 0){
			Settings::set_setting(count, tmpSettings);
			printf("%d\n", Settings::values[count]);
		}
		count++;
		current_index++;
	}
}

void network_thread(){
	printf("INFO: network_thread: started\n");

	std::string text;
	sockaddr_in sockName;
	sockaddr_in clientInfo; 
	int mainSocket;
	char buf[BUFSIZE];
	socklen_t addrlen;
	MESSAGE_TYPE response;
	
	mainSocket = socket(AF_INET, SOCK_STREAM, IPPROTO_TCP);
	if(mainSocket == -1)
		printf("ERROR: Couldn't create socket!\n");
	sockName.sin_family = AF_INET;
	sockName.sin_port =	htons(PORT);
	sockName.sin_addr.s_addr = INADDR_ANY;

	int yes = 1;
	if ( setsockopt(mainSocket, SOL_SOCKET, SO_REUSEADDR, &yes, sizeof(int)) == -1 )
	{
	    perror("setsockopt");
	}

	bind(mainSocket, (sockaddr*)&sockName, sizeof(sockName));
	listen(mainSocket, 10000000);
	while(!Settings::force_exit){
		
		addrlen = sizeof(clientInfo);
		client = accept(mainSocket, (sockaddr*)&clientInfo, &addrlen);
		cout << "INFO: Got a connection from " << inet_ntoa((in_addr)clientInfo.sin_addr) << endl;
		if (client != -1)
		 {
			 Settings::set_connected(true);
		 }

		while(Settings::connected && !Settings::force_exit){
			int msg_len = recv(client, buf, BUFSIZE - 1, 0);

			if (msg_len == -1)
			{
				printf("ERROR: Did not properly receive data.\n");
			}

			if(opt_debug)
				printf("DEBUG: Received %d bytes.\n", msg_len);

			response = parseMessage(buf);
			switch(response){
				case MSG_WAKEUP:
				{
					Settings::set_sleeping(false);
					Settings::set_initialized(true);
					break;
				}
				case MSG_SLEEP:
				{
					Settings::set_sleeping(true);
					Settings::set_initialized(false);
					break;
				}
				case MSG_SETTINGS:
				{
					if(!Settings::sleeping)
						printf("WARN: Can't change settings while the loop is running\n");
					else{
						changeSettings(buf);
						printf("INFO: Changed settings\n");
					}
					break;
				}
				case MSG_DISCONNECT:
				{
					Settings::set_connected(false);
					Settings::set_sleeping(true);
					Settings::set_initialized(false);
					break;
				}
				case MSG_REQUEST:
					Settings::set_requested_image(true);
					Settings::set_requested_type(BACKPROPAGATED);
					break;
				case MSG_REQUEST_RAW_G:
					Settings::set_requested_image(true);
					Settings::set_requested_type(RAW_G);
					break;
				case MSG_REQUEST_RAW_R:
					Settings::set_requested_image(true);
					Settings::set_requested_type(RAW_R);
					break;
				case MSG_COORDS:
					Settings::set_requested_coords(true);
					break;
				case MSG_HELLO:
				{
					send(client, "Hello!",7,0);
					break;
				}	
			} 
		}
		close(client);
	}
	close(mainSocket);

	printf("INFO: network_thread: ended\n");
}

void camera_thread(){
	printf("INFO: camera_thread: started\n");
	//Initializing LibArgus according to the tutorial for a sample project.
	// First we create a CameraProvider, necessary for each project.
	UniqueObj<CameraProvider> cameraProvider(CameraProvider::create());
	ICameraProvider* iCameraProvider = interface_cast<ICameraProvider>(cameraProvider);
	if(!iCameraProvider){
		printf("ERROR: Failed to establish libargus connection\n");
	}
	
	// Second we select a device from which to receive pictures (camera)
	std::vector<CameraDevice*> cameraDevices;
	iCameraProvider->getCameraDevices(&cameraDevices);
	if (cameraDevices.size() == 0){
		printf("ERROR: No camera devices available\n");
	}
	CameraDevice *selectedDevice = cameraDevices[0];

	// We create a capture session 
	UniqueObj<CaptureSession> captureSession(iCameraProvider->createCaptureSession(selectedDevice));
	ICaptureSession *iCaptureSession = interface_cast<ICaptureSession>(captureSession);
	if (!iCaptureSession){
 		printf("ERROR: Failed to create CaptureSession\n");
	}
	
	//CUDA variable declarations
	cudaEglStreamConnection conn;
	hipGraphicsResource_t resource;
	cudaEglFrame eglFrame;		
	hipArray_t yArray;
	hipArray_t uvArray;
	hipChannelFormatDesc yChannelDesc;
	hipChannelFormatDesc uvChannelDesc;

	while(!Settings::force_exit){
		while(Settings::connected && !Settings::force_exit){
			while(Settings::sleeping && Settings::connected && !Settings::force_exit){}
			if (Settings::force_exit) break;
			// Managing the settings for the capture session.
			UniqueObj<OutputStreamSettings> streamSettings(iCaptureSession->createOutputStreamSettings(STREAM_TYPE_EGL));
			IEGLOutputStreamSettings *iStreamSettings = interface_cast<IEGLOutputStreamSettings>(streamSettings);
			iStreamSettings->setPixelFormat(PIXEL_FMT_YCbCr_420_888);
			iStreamSettings->setResolution(Size2D<uint32_t>(WIDTH,HEIGHT));
			
			// Creating an Output stream. This should already create a producer.
			UniqueObj<OutputStream> outputStream(iCaptureSession->createOutputStream(streamSettings.get()));
			IEGLOutputStream *iEGLOutputStream = interface_cast<IEGLOutputStream>(outputStream);
            if (!iEGLOutputStream)
	            printf("Failed to create EGLOutputStream");

			eglStream = iEGLOutputStream->getEGLStream();
			cudaEGLStreamConsumerConnect(&conn, eglStream);
			
			// Managing requests.
			UniqueObj<Request> request(iCaptureSession->createRequest());
			IRequest *iRequest = interface_cast<IRequest>(request);
			iRequest->enableOutputStream(outputStream.get());
			
			ISourceSettings *iSourceSettings = interface_cast<ISourceSettings>(iRequest->getSourceSettings());
			iSourceSettings->setFrameDurationRange(Range<uint64_t>(1e9/DEFAULT_FPS));
			iSourceSettings->setExposureTimeRange(Range<uint64_t>(Settings::values[STG_EXPOSURE],Settings::values[STG_EXPOSURE]));
			iSourceSettings->setGainRange(Range<float>(0.5,1.5));

			IAutoControlSettings *iAutoSettings = interface_cast<IAutoControlSettings>(iRequest->getAutoControlSettings());
			iAutoSettings->setExposureCompensation(0);
			iAutoSettings->setIspDigitalGainRange(Range<float>(0,0));
			iAutoSettings->setWbGains(100);
			iAutoSettings->setColorSaturation(1.0);
			iAutoSettings->setColorSaturationBias(1.0);
			iAutoSettings->setColorSaturationEnable(true);
			iAutoSettings->setAwbLock(true);
			iAutoSettings->setAeAntibandingMode(AE_ANTIBANDING_MODE_OFF);

			IDenoiseSettings *iDenoiseSettings = interface_cast<IDenoiseSettings>(request);	
			iDenoiseSettings->setDenoiseMode(DENOISE_MODE_FAST);
			iDenoiseSettings->setDenoiseStrength(1.0);

			hipMalloc(&G, Settings::get_area()*sizeof(uint16_t));
			hipMalloc(&G_float, Settings::get_area()*sizeof(float));
			hipMalloc(&R, Settings::get_area()*sizeof(uint16_t));
			hipMalloc(&positionsGreen, Settings::get_area()*sizeof(float));
			hipMalloc(&positionsRed, Settings::get_area()*sizeof(float));		
			
			hipMalloc(&convolutionMaskGreen, CONVO_DIM_GREEN*CONVO_DIM_GREEN*sizeof(float));
			hipMalloc(&convolutionMaskRed, CONVO_DIM_RED*CONVO_DIM_RED*sizeof(float));
			numBlocks = 1024;
			generateConvoMaskGreen<<<numBlocks, BLOCKSIZE>>>(CONVO_DIM_GREEN, CONVO_DIM_GREEN, convolutionMaskGreen);
			generateConvoMaskRed<<<numBlocks, BLOCKSIZE>>>(CONVO_DIM_RED, CONVO_DIM_RED, convolutionMaskRed);
			
			hipMalloc(&kernelGreen, Settings::get_area()*sizeof(hipfftComplex));
			hipMalloc(&kernelRed, Settings::get_area()*sizeof(hipfftComplex));

			hipMalloc(&convolutionFilterBlur, Settings::get_area()*sizeof(hipfftComplex));
			generateBlurFilter<<<numBlocks, BLOCKSIZE>>>(dSTG_WIDTH, dSTG_HEIGHT, 3, convolutionFilterBlur);
			
			transformKernel(dSTG_WIDTH, dSTG_HEIGHT, CONVO_DIM_GREEN, convolutionMaskGreen, kernelGreen);
			transformKernel(dSTG_WIDTH, dSTG_HEIGHT, CONVO_DIM_RED, convolutionMaskRed, kernelRed);
			
			mtx.lock();
			hipMalloc(&maximaGreen, Settings::get_area()*sizeof(float));
			hipMalloc(&maximaRed, Settings::get_area()*sizeof(float));
			hipMalloc(&G_backprop, Settings::get_area()*sizeof(uint16_t));

			hipMalloc(&redOutputArray, Settings::get_area()*sizeof(float));
			mtx.unlock();

			yTexRef.normalized = 0;
			yTexRef.filterMode = hipFilterModePoint;
			yTexRef.addressMode[0] = hipAddressModeClamp;
			yTexRef.addressMode[1] = hipAddressModeClamp;
			hipGetTextureReference(&yTex, HIP_SYMBOL(&yTexRef));
			
			uvTexRef.normalized = 0;
			uvTexRef.filterMode = hipFilterModePoint;
			uvTexRef.addressMode[0] = hipAddressModeClamp;
			uvTexRef.addressMode[1] = hipAddressModeClamp;
			hipGetTextureReference(&uvTex, HIP_SYMBOL(&uvTexRef));

			// Initialize the BackPropagator for the green image
			BackPropagator backprop_G(dSTG_WIDTH, dSTG_HEIGHT, LAMBDA_GREEN, (float)Settings::values[STG_Z_GREEN]/(float)1000000);
			
			//CUDA initialization
			//Main loop
			auto initializer = std::chrono::system_clock::now();
			std::chrono::duration<double> elapsed_seconds_average = initializer-initializer;

			final_count = 0;
			while(!Settings::initialized && Settings::connected && !Settings::force_exit){}
			if (Settings::force_exit) break;

			while(!Settings::sleeping && Settings::connected && ! Settings::force_exit){
				auto start = std::chrono::system_clock::now();
				
				
				iCaptureSession->capture(request.get());
				res = cudaEGLStreamConsumerAcquireFrame(&conn, &resource, 0, 5000);
				if(res != hipSuccess){
					continue;
				}
				cudaGraphicsResourceGetMappedEglFrame(&eglFrame, resource, 0, 0);
				yArray = eglFrame.frame.pArray[0];
				uvArray = eglFrame.frame.pArray[1];
				
				hipGetChannelDesc(&yChannelDesc, (hipArray_const_t)(yArray));
				hipBindTextureToArray(yTex, (hipArray_const_t)(yArray), &yChannelDesc);
				hipGetChannelDesc(&uvChannelDesc, (hipArray_const_t)(uvArray));
				hipBindTextureToArray(uvTex, (hipArray_const_t)(uvArray), &uvChannelDesc);
				auto initialization = std::chrono::system_clock::now();

				numBlocks = (Settings::get_area()/2 +BLOCKSIZE -1)/BLOCKSIZE;
				
				mtx.lock();
				yuv2bgr<<<numBlocks, BLOCKSIZE>>>(dSTG_WIDTH, dSTG_HEIGHT,
												Settings::values[STG_OFFSET_X], Settings::values[STG_OFFSET_Y], G, R);
				backprop_G.backprop(G, G_backprop);

				u16ToFloat<<<numBlocks, BLOCKSIZE>>>(dSTG_WIDTH, dSTG_HEIGHT, G_backprop, G_float);

				mtx.unlock();
				
				auto test2 = std::chrono::system_clock::now();
				std::chrono::duration<double> elapsed_seconds = test2-initialization;
				
				if(opt_verbose) {
					std::cout << "TRACE: Converting the image format + backprop took: " << elapsed_seconds.count() << "s\n";
				}

				hipUnbindTexture(yTex);
				hipUnbindTexture(uvTex);
				
				cudaEGLStreamConsumerReleaseFrame(&conn, resource, 0);
				
				auto end = std::chrono::system_clock::now();
				elapsed_seconds = end-start;
				elapsed_seconds_average +=elapsed_seconds;
				final_count++;
				Settings::sent_coords = false;
				
				if(opt_verbose) {
					std::cout << "TRACE: This cycle took: " << elapsed_seconds.count() << "s\n";
				}

				cycles++;				
			}
			std::cout << "INFO: Average time to complete a cycle: " << elapsed_seconds_average.count()/final_count << "s\n";
			iCaptureSession->waitForIdle();
			
			hipFree(G);
			hipFree(R);
			hipFree(G_backprop);
			hipFree(redOutputArray);
			hipFree(maximaGreen);
			hipFree(maximaRed);
			hipFree(convolutionMaskGreen);
			hipFree(convolutionMaskRed);
			hipFree(kernelGreen);
			hipFree(kernelRed);
			
			cudaEGLStreamConsumerDisconnect(&conn);
			iEGLOutputStream->disconnect();
			outputStream.reset();
		}
	}

	printf("INFO: camera_thread: ended\n");
}

void mouseEventCallback(int event, int x, int y, int flags, void* userdata)
{
	// https://www.opencv-srf.com/2011/11/mouse-events.html
	if ( event == CV_EVENT_MOUSEMOVE )
     {
     	if(opt_debug)
        	cout << "DEBUG: Mouse move over the window - position (" << x << ", " << y << ")" << endl;
        Settings::set_force_exit(true);
     }
}

void datasend_thread(){
	printf("INFO: datasend_thread: started\n");

	float *temporary;
	float *temporary_red_positions;
	float *temporary_green_positions;
	char* buffer;
	while(true){
		while(Settings::sleeping && !Settings::force_exit){}
		if (Settings::force_exit) break;

		hipMalloc(&temporary, Settings::get_area()*sizeof(float));
		hipMalloc(&temporary_red_positions, Settings::get_area()*sizeof(float));
		hipMalloc(&temporary_green_positions, Settings::get_area()*sizeof(float));
		
		while(Settings::connected && !Settings::sleeping){
			if(Settings::requested_image){
				mtx.lock();
				switch (Settings::requested_type){
					case BACKPROPAGATED:
						hipMemcpy(temporary, G_backprop, sizeof(float)*Settings::get_area(), hipMemcpyDeviceToDevice);
						break;
					case RAW_G:
						hipMemcpy(temporary, G, sizeof(float)*Settings::get_area(), hipMemcpyDeviceToDevice);
						break;
					case RAW_R:
						hipMemcpy(temporary, R, sizeof(float)*Settings::get_area(), hipMemcpyDeviceToDevice);
						break;
				}	

				mtx.unlock();

				buffer = (char*)malloc(Settings::get_area()*sizeof(float));
				hipMemcpy(buffer, temporary, sizeof(float)*Settings::get_area(), hipMemcpyDeviceToHost);
				send(client, buffer, sizeof(float)*Settings::get_area(), 0);
				free(buffer);
				printf("INFO: Image sent.\n");
				Settings::set_requested_image(false);
			}
			if(!Settings::sent_coords && Settings::requested_coords){
				int* sorted_green_positions = (int*)malloc(sizeof(int)*Settings::get_area());
				int* sorted_red_positions = (int*)malloc(sizeof(int)*Settings::get_area());
				mtx.lock();
				hipMemcpy(temporary_green_positions, maximaGreen, sizeof(int)*Settings::get_area(), hipMemcpyDeviceToDevice);
				hipMemcpy(temporary_red_positions, maximaRed, sizeof(int)*Settings::get_area(), hipMemcpyDeviceToDevice);
				mtx.unlock();

				int* count = (int*)malloc(sizeof(int)*2);

				processPoints(temporary_green_positions, temporary_red_positions, sorted_green_positions, sorted_red_positions, count);

				buffer = (char*)malloc(sizeof(int)*(2+count[0]+count[1]));

				if(opt_debug)
					printf("DEBUG: Count Green : %d ; Count Red : %d\n", count[0], count[1]);

				memcpy(&buffer[0], &count[0], sizeof(int));
				memcpy(&buffer[4], sorted_green_positions, count[0]*sizeof(int));
				memcpy(&buffer[4*(1+count[0])], &count[1], sizeof(int));
				memcpy(&buffer[4*(2+count[0])], sorted_red_positions, count[1]*sizeof(int));

				send(client, buffer, sizeof(int)*(2+count[0]+count[1]), 0);

				printf("INFO: Sent the found locations\n");

				free(buffer);
				free(count);
				free(sorted_green_positions);
				free(sorted_red_positions);

				Settings::set_sent_coords(true);
				Settings::set_requested_coords(false);
			}

			if (Settings::force_exit) break;
		}
		hipFree(temporary_red_positions);
		hipFree(temporary_green_positions);
		
		hipFree(temporary);
	}

	printf("INFO: datasend_thread: ended\n");
}

void display_thread(){
	printf("INFO: display_thread: started\n");

	float* imageToDisplay;
	char ret_key;
	char filename [50];

	while(true){
		while(Settings::sleeping && Settings::connected && !Settings::force_exit){}
		if (Settings::force_exit) break;
			
		hipMalloc(&imageToDisplay, sizeof(float)*Settings::get_area());
		float* output = (float*)malloc(sizeof(float)*Settings::get_area());
		cv::namedWindow("Basic Visualization", CV_WINDOW_NORMAL);
		cv::setWindowProperty("Basic Visualization", CV_WND_PROP_FULLSCREEN, CV_WINDOW_FULLSCREEN);
		//set the callback function for any mouse event
		if (opt_mousekill) {
			 cv::setMouseCallback("Basic Visualization", mouseEventCallback, NULL);
		}

		while(!Settings::initialized && Settings::connected && !Settings::force_exit){}
		if (Settings::force_exit){
			hipFree(imageToDisplay);
			free(output);
			break;
		} 

		const cv::Mat img_trans(cv::Size(dSTG_WIDTH, dSTG_HEIGHT), CV_32F);		
		const cv::Mat img_u8(cv::Size(dSTG_WIDTH, dSTG_HEIGHT), CV_8U);

		while(!Settings::sleeping && Settings::connected){
			if(cycles >= 3){
				auto start = std::chrono::system_clock::now();
				cycles = 0;
				mtx.lock();
				hipMemcpy(imageToDisplay, G_float, sizeof(float)*Settings::get_area(), hipMemcpyDeviceToDevice);
				// hipMemcpy(imageToDisplay, G_backprop, sizeof(float)*Settings::get_area(), hipMemcpyDeviceToDevice);
				mtx.unlock();

				hipMemcpy(output, imageToDisplay, sizeof(float)*Settings::get_area(), hipMemcpyDeviceToHost);
				const cv::Mat img(cv::Size(dSTG_WIDTH, dSTG_HEIGHT), CV_32F, output);

				if (opt_saveimgs) {
					sprintf (filename, "./imgs/img_%05d.png", final_count);
					cv::imwrite( filename, img );
				} else {
					// Flip the image only if the images are not stored (to save some time)
					cv::flip(img, img_trans, -1);
					cv::transpose(img_trans, img);
					
					img.convertTo(img_u8,CV_8U);
				}

				cv::imshow("Basic Visualization", img_u8);
				auto end = std::chrono::system_clock::now();
				std::chrono::duration<double> elapsed_seconds = end-start;if(opt_verbose) {
					std::cout << "TRACE: Stroring the image took: " << elapsed_seconds.count() << "s\n";
				}

				ret_key = (char) cv::waitKey(1);
				if (ret_key == 27 || ret_key == 'x') Settings::set_force_exit(true);  // exit the app if `esc' or 'x' key was pressed.
			}
			else{
				usleep(5000);
			}

			if (Settings::force_exit) break;
		}
		hipFree(imageToDisplay);
		free(output);
	}

	printf("INFO: display_thread: ended\n");
}


int main(int argc, char* argv[]){
	if(opt_debug){
		printf("DEBUG: Initial settings:");
		for(int i = 0 ; i < STG_NUMBER_OF_SETTINGS; i++){
			printf("%d\n", Settings::values[i]);
		}
	}
  	auto result = parse(argc, argv);
	
	cycles = 0;

	if (opt_show) {
		Settings::set_initialized(true);
		Settings::set_connected(true);
		Settings::set_sleeping(false);
	}

	thread camera_thr (camera_thread);
	thread display_thr (display_thread);
	thread network_thr (network_thread);
	thread datasend_thr (datasend_thread);
	thread keyboard_thr (keyboard_thread);
	
	camera_thr.join();
	display_thr.join();
	datasend_thr.join();

	return 0;
}
