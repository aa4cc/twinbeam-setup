#include "hip/hip_runtime.h"
/**
 * @author  Viktor-Adam Koropecky
 */

#include "stdint.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "stdio.h"
#include "Kernels.h"
#include "math.h"
#include <cmath>

/*
    Calculation of the Hq matrix according to the equations in original .m file
*/
__global__ void calculate(int N, int M, float z, float dx, float n, float lambda, hipfftComplex* Hq)
        {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            float FX, FY, temp, res;
            float pre = n/lambda;
            float calc = 1/dx;
            int newIndex;
            int count = N*M;
            for (int i = index; i < count; i += stride)
            {
                newIndex = (i + count/2-1) % (count);
                FX = ((float)(1+(i/M)) * calc/(float)(N)) - calc/2.0f;
                FY = ((float)(1+(i%M)) * calc/(float)(M)) - calc/2.0f;
                res = 2 * PI*z*pre * sqrt(1 - SQUARE(FX/pre) - SQUARE(FY/pre));
                //temp = (sqrt(SQUARE(FX) + SQUARE(FY)) < (pre));
                if(temp == 0.0){
					Hq[(newIndex % M) > M/2-1 ? newIndex-M/2 : newIndex+M/2] = make_hipComplex(0,0);
				}
				else{
					Hq[(newIndex % M) > M/2-1 ? newIndex-M/2 : newIndex+M/2] = make_hipComplex(std::cos(res),std::sin(res));
				}
            }
        }

/*
    Element-wise multiplication of two (already transposed) matrices.
*/
__global__ void elMultiplication(int N, int M, hipfftComplex*  Hq, hipfftComplex*  Bq){
            hipfftComplex temp;
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for(int i = index; i < N*M; i += stride){
                temp = make_hipFloatComplex(Bq[i].x/(float)(N*M), Bq[i].y/(float)(N*M));
                Bq[i] = hipCmulf(Hq[i], temp);
            }
        }
        
__global__ void elMultiplication2(int N, int M, hipfftComplex*  input, hipfftComplex*  kernel, hipfftComplex* output){
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for(int i = index; i < N*M; i += stride){
                output[i] = hipCmulf(kernel[i], input[i]);
            }
        }

/*
    Converting a complex hipfftComplex array to a float array of its absolute values
*/
__global__ void absoluteValue(int N, int M, hipfftComplex* storageArray, float* outputArray){
            hipfftComplex temp;
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for(int i = index; i < N*M; i += stride){
                //temp = make_hipFloatComplex(storageArray[(i%N)*M + i/N].x, storageArray[i%N*M + i/N].y);
                temp = make_hipFloatComplex(storageArray[i].x, storageArray[i].y);
                outputArray[i] = hipCabsf(temp);
            }
        }
        
__global__ void cutAndConvert(int N, int M, hipfftComplex* input, float* output){
            hipfftComplex temp;
            float floatTemp;
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for(int i = index; i < N*M; i += stride){
                floatTemp = hipCabsf(input[i]);
                if(floatTemp > 0 && input[i].x+input[i].y > 0)
					output[i] = floatTemp;
				else
					output[i] = 0;
            }
        }

/*
    Transposition of a float matrix
*/
__global__ void transpose(int N, int M, float* input, float* output){
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for(int i = index; i < N*M; i += stride){
                output[i] = input[(i%N)*M + i/N];
            }
        }

__global__ void u16ToDouble(int N, int M, uint16_t* transposee, float* result){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < N*M; i += stride){
        result[i] = (float)transposee[i];
    }
}

/*
    Converting a real mxArray to complex hipfftComplex array
*/
__global__ void convertToComplex(int count , float* real, hipfftComplex* complex){
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for(int i = index; i < count; i += stride){
                complex[i] = make_hipComplex(real[i], 0);
            }
        }

__global__ void convertToFloat(int count , float* output, hipfftComplex* input){
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for(int i = index; i < count; i += stride){
                output[i] = hipCabsf(input[i]);
            }
        }

/*
    Bayerization function to transform camera data to a Bayer picture
*/
__global__ void bayerize(int M, int N, uint8_t* input, uint16_t* output){
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            int count = N*M;
            int temp;
            for(int i = index; i < count; i += stride){
                temp = ((i/M) % 2) == 1 ? count*2 : count;
                output[i] = (uint16_t)(input[i + temp])*eighth_power(2) + (uint16_t)(input[i]);
            }
        }

/*
    Debayerization function to transform input data into an RGB image.

    Bayer pattern is bggr or 

    B | G | B | G  
    -   -   -   - 
    G | R | G | R
    -   -   -   -
    B | G | B | G 
    -   -   -   - 
    G | R | G | R

    from which we want to demosaic an rbg bitmap picture

    R G B | R G B | R G B | R G B  
    -----   -----   -----   -----
    R G B | R G B | R G B | R G B  
    -----   -----   -----   -----
    R G B | R G B | R G B | R G B  
    -----   -----   -----   ----- 
    R G B | R G B | R G B | R G B  
*/
__global__ void demosaic(int M, int N, uint16_t* input, uint16_t* R, uint16_t* G, uint16_t* B){
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            int count = N*M;
            bool evenColumn;
            bool evenRow;
            bool firstColumn, lastColumn;
            bool firstRow, lastRow;
            for(int i = index; i < count; i += stride){
                evenColumn = (((i/M) % 2) == 1) ? true : false;
                evenRow = (((i%M) % 2) == 1) ? true : false;
                firstColumn = ((i/M) == 0) ? true : false;
                lastColumn = ((i/M) == (N-1)) ? true : false;
                firstRow = ((i&M) == 0) ? true : false;
                lastRow = ((i&M) == (M-1)) ? true : false;
                if(evenColumn){
                    if(evenRow){
                        R[i] = input[i];
                        if(lastColumn){
                            if(lastRow){
                                B[i] = input[i-M-1];
                                G[i] = (input[i-1] + input[i-M])/2;
                            }
                            else{
                                B[i] = (input[i-M-1] + input[i-M+1])/2;
                                G[i] = (input[i-1] + input[i+1])/2;
                            }
                        }
                        else{
                            if(lastRow){
                                B[i] = (input[i-M-1] + input[i+M-1])/2;
                                G[i] = (input[i-M] + input[i+M])/2;
                            }
                            else{
                                B[i] = (input[i-M-1] + input[i+M+1] + input[i-M+1] + input[i+M-1])/4;
                                G[i] = (input[i-1] + input[i+1] + input[i-M] + input[i+M])/4;
                            }  
                        }
                    }
                    else{
                        G[i] = input[i];
                        if(firstRow){
                            R[i] = input[i+1];
                        }
                        else{
                            R[i] = (input[i+1] + input[i-1])/2;
                        }
                        if(lastColumn){
                            B[i] = input[i-M];
                        }
                        else{
                            B[i] = (input[i+M] + input[i-M])/2;
                        }
                    }
                }
                else{
                    if(evenRow){
                        G[i] = input[i];
                        if(lastRow){
                            B[i] = input[i-1];
                        }
                        else{
                            B[i] = (input[i+1] + input[i-1])/2;
                        }
                        if(firstColumn){
                            R[i] = input[i+M];
                        }
                        else{
                            R[i] = (input[i+M] + input[i-M])/2;
                        }
                    }
                    else{
                        B[i] = input[i];
                        if(firstColumn){
                            if(firstRow){
                                R[i] = input[i+M+1];
                                G[i] = (input[i+1] + input[i+M])/2;
                            }
                            else{
                                R[i] = (input[i+M+1] + input[i+M-1])/2;
                                G[i] = (input[i+1] + input[i-1])/2;
                            }
                        }
                        else{
                            if(firstRow){
                                R[i] = (input[i+M+1] + input[i-M+1])/2;
                                G[i] = (input[i-M] + input[i+M])/2;
                            }
                            else{
                                R[i] = (input[i-M-1] + input[i+M+1] + input[i-M+1] + input[i+M-1])/4;
                                G[i] = (input[i-1] + input[i+1] + input[i-M] + input[i+M])/4;
                            }  
                        }
                    }
                }
            }
        }

//Function that returns desampled 
__global__ void desample(int M, int N, float* input, float* output){
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            int count = N*M/4;
            for(int i = index; i < count; i += stride){
                output[i] = input[i*2 + M*((i*2)/M) ];
                //lol
            }
}


__global__ void generateConvoMaskRed(int m, int n, float* convoMask){
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            int count = m*n;
            double temp;
            for(int i = index; i < count; i += stride){
                temp = sqrt((double)(SQUARE((double)((i%m) - (double)(m/2))) + SQUARE((double)((i/m) - (double)(m/2)))));
                convoMask[i] = 0;
                if( temp <= 24 ){
                    convoMask[i] = -0.5;
                }
                else if(temp > 27 && temp <= 30){
                    convoMask[i] = 1.3;
                }
            }
}

//This convolution core returns so far the best results for the green chanel. 
__global__ void generateConvoMaskGreen(int m, int n, float* convoMask){
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            int count = m*n;
            double temp;
            for(int i = index; i < count; i += stride){
                temp = sqrt((double)(SQUARE((double)((i%m) - (double)(m/2))) + SQUARE((double)((i/m) - (double)(m/2)))));
                convoMask[i] = 0;
                if( temp <= 23 ){
                    convoMask[i] = -0.78;
                }
                else if(temp > 23 && temp <= 28){
                    convoMask[i] = 1;
                }
            }
}

__global__ void sobelDerivation(int M, int N, float* input, float* output){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int count = M*N;
    int xFilter[9] = {1 , 2 , 1 , 0 , 0 , 0 , -1 , -2 , -1};
    int yFilter[9] = {1 , 0 , -1 , 2 , 0 , -2 , 1 , 0 , -1};
    short filterSize = 3;
    float xTemp;
    float yTemp;
    for(int i = index; i < count; i += stride){
        xTemp = 0;
        yTemp = 0;
        output[i] = 0;
        if( (M - i%M) > filterSize && (N - i/M) > filterSize ){
            for(int j = 0; j < SQUARE(filterSize); j++){
                xTemp += xFilter[j] * input[i + j%filterSize + M*(j/filterSize)];
                yTemp += yFilter[j] * input[i + j%filterSize + M*(j/filterSize)];
            }
            output[i] = sqrt(SQUARE(xTemp) + SQUARE(yTemp));

        }
    }
}

__global__ void findExtremes(int M, int N, float* input, float* extremes){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int count = M*N;
    for(int i = index; i < count; i += stride){
        if(input[i] > extremes[0])
            extremes[0] = input[i];
    } 
}

__global__ void normalize(int M, int N, float* input, float* extremes){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int count = M*N;
    for(int i = index; i < count; i += stride){
        input[i] = input[i] / extremes[0];
    } 
}

__global__ void getLocalMaxima(int M, int N, float* input, float* output){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int count = M*N;
    bool passable;
    for(int i = index; i < count; i += stride){
        passable = true;
        output[i] = 0;
        if( i % M != 0 && input[i-1] > input[i] )
            passable = false;
        if( i / M != 0 && input[i-M] > input[i] )
            passable = false;
        if( i % M != M-1 && input[i+1] > input[i] )
            passable = false;
        if( i / M != M-1 && input[i+M] > input[i] )
            passable = false;
        if( passable == true )
            output[i] = input[i];
    }
}

__global__ void kernelToImage(int M, int N, int kernelDim, float* kernel, hipfftComplex* outputKernel){
			int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            int count = kernelDim*kernelDim;
            int center = (kernelDim)/2;
            for(int i = index; i < count; i += stride){
				int lineNum = i/kernelDim;
                int colNum = i%kernelDim;
                if(colNum >= center && lineNum >= center)
                    outputKernel[colNum-center + (lineNum-center)*M] = make_hipComplex(kernel[i], 0);
                else if(colNum >= center)
                    outputKernel[colNum-center + (N - center + lineNum)*M] = make_hipComplex(kernel[i], 0);
                else if(lineNum >= center)
                    outputKernel[colNum + M - center + (lineNum-center)*M] = make_hipComplex(kernel[i], 0);
                else
                    outputKernel[colNum + M - center + (N - center + lineNum)*M] = make_hipComplex(kernel[i], 0);
            }
	}
	
__global__ void findPoints(int M, int N, float* input, int* output){
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		int stride = blockDim.x * gridDim.x;
		int count = N*M;
		for(int i = index; i < count; i += stride){
			if(input[i] > 0){
				output[i] = i;
			}
		}
	
	}

/*
__global__ void stupidSort(int M, int N, int* input, int* output, int *currentIndex){
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		int stride = blockDim.x * gridDim.x;
		int count = N*M;
		for(int i = index; i < count; i += stride){
			if(input[i] > 0){
				atomicAdd(currentIndex[0], 1); 
				output[currentIndex[0]] = input[i];
			}
		}
	
	}

*/