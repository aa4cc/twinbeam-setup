#include "hip/hip_runtime.h"
/**
 * @author  Viktor-Adam Koropecky
 */

#include "stdint.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "stdio.h"
#include "Kernels.h"
#include "math.h"
#include <cmath>
#include "Definitions.h"

/*
    Calculation of the Hq matrix according to the equations in original .m file
*/
__global__ void calculate(int N, int M, float z, float dx, float n, float lambda, hipfftComplex* Hq)
        {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            float FX, FY, temp, res;
            float pre = n/lambda;
            float calc = 1/dx;
            int newIndex;
            int count = N*M;
            for (int i = index; i < count; i += stride)
            {
                newIndex = (i + count/2-1) % (count);
                FX = ((float)(1+(i/M)) * calc/(float)(N)) - calc/2.0f;
                FY = ((float)(1+(i%M)) * calc/(float)(M)) - calc/2.0f;
                res = 2 * PI*z*pre * sqrt(1 - SQUARE(FX/pre) - SQUARE(FY/pre));
                //temp = (sqrt(SQUARE(FX) + SQUARE(FY)) < (pre));
                if(temp == 0.0){
					Hq[(newIndex % M) > M/2-1 ? newIndex-M/2 : newIndex+M/2] = make_hipComplex(0,0);
				}
				else{
					Hq[(newIndex % M) > M/2-1 ? newIndex-M/2 : newIndex+M/2] = make_hipComplex(std::cos(res),std::sin(res));
				}
            }
        }

/*
    Element-wise multiplication of two (already transposed) matrices.
*/
__global__ void elMultiplication(int N, int M, hipfftComplex*  Hq, hipfftComplex*  Bq){
            hipfftComplex temp;
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for(int i = index; i < N*M; i += stride){
                temp = make_hipFloatComplex(Bq[i].x/(float)(N*M), Bq[i].y/(float)(N*M));
                Bq[i] = hipCmulf(Hq[i], temp);
            }
        }
        
__global__ void elMultiplication2(int N, int M, hipfftComplex*  input, hipfftComplex*  kernel, hipfftComplex* output){
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for(int i = index; i < N*M; i += stride){
                output[i] = hipCmulf(kernel[i], input[i]);
            }
        }

/*
    Converting a complex hipfftComplex array to a float array of its absolute values
*/
__global__ void absoluteValue(int N, int M, hipfftComplex* storageArray, float* outputArray){
            hipfftComplex temp;
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for(int i = index; i < N*M; i += stride){
                //temp = make_hipFloatComplex(storageArray[(i%N)*M + i/N].x, storageArray[i%N*M + i/N].y);
                temp = make_hipFloatComplex(storageArray[i].x, storageArray[i].y);
                outputArray[i] = hipCabsf(temp);
            }
        }
        
__global__ void cutAndConvert(int N, int M, hipfftComplex* input, float* output){
            float floatTemp;
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for(int i = index; i < N*M; i += stride){
                floatTemp = hipCabsf(input[i]);
                if(floatTemp > 0 && input[i].x+input[i].y > 0)
					output[i] = floatTemp;
				else
					output[i] = 0;
            }
        }

/*
    Transposition of a float matrix
*/
__global__ void transpose(int N, int M, float* input, float* output){
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for(int i = index; i < N*M; i += stride){
                output[i] = input[(i%N)*M + i/N];
            }
        }

__global__ void u16ToDouble(int N, int M, uint16_t* transposee, float* result){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < N*M; i += stride){
        result[i] = (float)transposee[i];
    }
}

/*
    Converting a real mxArray to complex hipfftComplex array
*/
__global__ void convertToComplex(int count , float* real, hipfftComplex* complex){
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for(int i = index; i < count; i += stride){
                complex[i] = make_hipComplex(real[i], 0);
            }
        }

__global__ void convertToFloat(int count , float* output, hipfftComplex* input){
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for(int i = index; i < count; i += stride){
                output[i] = hipCabsf(input[i]);
            }
        }

//Function that returns desampled 
__global__ void desample(int M, int N, float* input, float* output){
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            int count = N*M/4;
            for(int i = index; i < count; i += stride){
                output[i] = input[i*2 + M*((i*2)/M) ];
                //lol
            }
}


__global__ void generateConvoMaskRed(int m, int n, float* convoMask){
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            int count = m*n;
            double temp;
            for(int i = index; i < count; i += stride){
                temp = sqrt((double)(SQUARE((double)((i%m) - (double)(m/2))) + SQUARE((double)((i/m) - (double)(m/2)))));
                convoMask[i] = 0;
                if( temp <= 24 ){
                    convoMask[i] = -0.5;
                }
                else if(temp > 27 && temp <= 30){
                    convoMask[i] = 1.3;
                }
            }
}

//This convolution core returns so far the best results for the green chanel. 
__global__ void generateConvoMaskGreen(int m, int n, float* convoMask){
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            int count = m*n;
            double temp;
            for(int i = index; i < count; i += stride){
                temp = sqrt((double)(SQUARE((double)((i%m) - (double)(m/2))) + SQUARE((double)((i/m) - (double)(m/2)))));
                convoMask[i] = 0;
                if( temp <= 23 ){
                    convoMask[i] = -0.78;
                }
                else if(temp > 23 && temp <= 28){
                    convoMask[i] = 1;
                }
            }
}

__global__ void sobelDerivation(int M, int N, float* input, float* output){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int count = M*N;
    int xFilter[9] = {1 , 2 , 1 , 0 , 0 , 0 , -1 , -2 , -1};
    int yFilter[9] = {1 , 0 , -1 , 2 , 0 , -2 , 1 , 0 , -1};
    short filterSize = 3;
    float xTemp;
    float yTemp;
    for(int i = index; i < count; i += stride){
        xTemp = 0;
        yTemp = 0;
        output[i] = 0;
        if( (M - i%M) > filterSize && (N - i/M) > filterSize ){
            for(int j = 0; j < SQUARE(filterSize); j++){
                xTemp += xFilter[j] * input[i + j%filterSize + M*(j/filterSize)];
                yTemp += yFilter[j] * input[i + j%filterSize + M*(j/filterSize)];
            }
            output[i] = sqrt(SQUARE(xTemp) + SQUARE(yTemp));

        }
    }
}

__global__ void findExtremes(int M, int N, float* input, float* extremes){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int count = M*N;
    for(int i = index; i < count; i += stride){
        if(input[i] > extremes[0])
            extremes[0] = input[i];
    } 
}

__global__ void normalize(int M, int N, float* input, float* extremes){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int count = M*N;
    for(int i = index; i < count; i += stride){
        input[i] = input[i] / extremes[0];
    } 
}

__global__ void getLocalMaxima(int M, int N, float* input, float* output){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int count = M*N;
    bool passable;
    for(int i = index; i < count; i += stride){
        passable = true;
        output[i] = 0;
        if( i % M != 0 && input[i-1] > input[i] )
            passable = false;
        if( i / M != 0 && input[i-M] > input[i] )
            passable = false;
        if( i % M != M-1 && input[i+1] > input[i] )
            passable = false;
        if( i / M != M-1 && input[i+M] > input[i] )
            passable = false;
        if( passable == true )
            output[i] = input[i];
    }
}

__global__ void kernelToImage(int M, int N, int kernelDim, float* kernel, hipfftComplex* outputKernel){
			int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            int count = kernelDim*kernelDim;
            int center = (kernelDim)/2;
            for(int i = index; i < count; i += stride){
				int lineNum = i/kernelDim;
                int colNum = i%kernelDim;
                if(colNum >= center && lineNum >= center)
                    outputKernel[colNum-center + (lineNum-center)*M] = make_hipComplex(kernel[i], 0);
                else if(colNum >= center)
                    outputKernel[colNum-center + (N - center + lineNum)*M] = make_hipComplex(kernel[i], 0);
                else if(lineNum >= center)
                    outputKernel[colNum + M - center + (lineNum-center)*M] = make_hipComplex(kernel[i], 0);
                else
                    outputKernel[colNum + M - center + (N - center + lineNum)*M] = make_hipComplex(kernel[i], 0);
            }
	}
	
__global__ void findPoints(int M, int N, float* input, int* output, int* counter){
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		int stride = blockDim.x * gridDim.x;
		int count = N*M;
		for(int i = index; i < count; i += stride){
			if(input[i] > 0){
				output[i] = i;
				counter[0] += 1;
			}
		}
	
	}

__global__ void stupidSort(int M, int N, int* input, int* output, int *currentIndex){
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		int stride = blockDim.x * gridDim.x;
		int count = N*M;
		for(int i = index; i < count; i += stride){
			if(input[i] > 0){
				atomicAdd(currentIndex, 1); 
				output[*currentIndex] = input[i];
			}
		}
	
	}

