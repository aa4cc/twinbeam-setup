#include "ColorChannel.h"
#include "Settings.h"
#include "Definitions.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "Kernels.h"

int numBlocks = (Settings::get_area()/2 +BLOCKSIZE -1)/BLOCKSIZE; 

void ColorChannel::initialize(bool d, int z, float l){
	display = d;
	z = zi;
	lambda = l;
}

void ColorChannel::allocate(){
	hipMalloc(&original, Settings::get_area()*sizeof(uint16_t));
	hipMalloc(&doubleOriginal, Settings::get_area()*sizeof(float));
	hipMalloc(&maxima, Settings::get_area()*sizeof(float));
	hipMalloc(&hq, Settings::get_area()*sizeof(hipfftComplex));
	hipMalloc(&convoluted, Settings::get_area()*sizeof(hipfftComplex));
	calculateHq();

	if(display){
		hipMalloc(&backpropagated, Settings::get_area()*sizeof(float));
	}
}

void ColorChannel::deallocate(){
	hipFree(original);
	hipFree(doubleOriginal);
	hipFree(maxima);
	hipFree(hq);
	hipFree(convoluted);

	if(display){
		hipFree(backpropagated);
	}
}

void ColorChannel::calculateHq(){
	calculate<<<numBlocks, BLOCKSIZE>>>(STG_HEIGHT, STG_WIDTH, z, PIXEL_DX, REFRACTION_INDEX, lambda, hq);
}

void ColorChannel::typeCast(){
	u16ToDouble<<<numBlocks, BLOCKSIZE>>>(STG_HEIGHT, STG_WIDTH, original, doubleOriginal);
}

void ColorChannel::backpropagate(hipfftComplex* kernel){
	hipfftComplex* image;
    hipfftComplex* convolutedImage;
    float* filterOutput;
    float* extremes;
    hipfftHandle t_plan;

    hipMalloc(&filterOutput, Settings::get_area()*sizeof(float));
    hipMalloc(&extremes, Settings::get_area()*sizeof(float));
    hipMalloc(&image, Settings::get_area()*sizeof(hipfftComplex));
    hipMalloc(&convolutedImage, 2*sizeof(hipfftComplex));

    convertToComplex<<<numBlocks, BLOCKSIZE>>>(Settings::get_area(), doubleOriginal, image);
    hipfftPlan2d(&plan, STG_HEIGHT, STG_WIDTH, HIPFFT_C2C);
    hipfftExecC2C(plan, image, image, HIPFFT_FORWARD);
	multiplyInPlace<<<numBlocks, BLOCKSIZE>>>(STG_WIDTH, STG_HEIGHT, hq, image);
	multiply<<<numBlocks, BLOCKSIZE>>>(STG_WIDTH, STG_HEIGHT, image, kernel, convolutedImage);
	
	if(display){
		hipfftExecC2C(plan, image, image, HIPFFT_BACKWARD);
		absoluteValue<<<numBlocks, BLOCKSIZE>>>(STG_WIDTH, STG_HEIGHT, image, backpropagated);
		findExtremes<<<numBlocks, BLOCKSIZE>>>(STG_WIDTH, STG_HEIGHT, backpropagated, extremes);
		normalize<<<numBlocks, BLOCKSIZE>>>(STG_WIDTH, STG_HEIGHT, backpropagated, extremes);
	}

	//Current version of object position detection, needs to be updated

	hipfftExecC2C(plan, convolutedImage, convolutedImage, HIPFFT_BACKWARD);
	cutAndConvert<<<numBlocks, BLOCKSIZE>>>(STG_WIDTH, STG_HEIGHT, convolutedImage, maxima);

    hipFree(extremes);
    hipMalloc(&extremes, sizeof(float)*2);

	findExtremes<<<numBlocks, BLOCKSIZE>>>(STG_WIDTH, STG_HEIGHT, filterOutput, extremes);
	normalize<<<numBlocks, BLOCKSIZE>>>(STG_WIDTH, STG_HEIGHT, filterOutput, extremes);
	getLocalMaxima<<<numBlocks, BLOCKSIZE>>>(STG_WIDTH, STG_HEIGHT, filterOutput, maxima);

	hipfftDestroy(plan);
    hipFree(extremes);
    hipFree(convolutedImage);
    hipFree(image);
    hipFree(filterOutput);


}
