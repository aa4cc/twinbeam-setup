/**
 * @author  Martin Gurtner
 */

#include "ImageData.h"

template<typename T>
bool ImageData<T>::create(uint16_t m, uint16_t n) {
    release();
    width = m; height = n;
    // Allocate memory on the host side
    if(hipHostAlloc((void **)&h_data,  sizeof(T)*width*height,  hipHostMallocMapped) != hipSuccess) return false;
    // Allocate memory on the device side
    if(hipMalloc((void **)&d_data,  sizeof(T)*width*height) != hipSuccess) return false;

    return true;
};

template<typename T>
void ImageData<T>::release() {
    if (h_data) hipHostFree(h_data);
    if (d_data) hipFree(d_data);
};

template<typename T>
T* ImageData<T>::hostPtr(bool sync) {
    if(sync) {
        std::lock_guard<std::mutex> l_src(mtx);

        hipMemcpy(h_data, d_data, sizeof(T)*width*height, hipMemcpyDeviceToHost);
    }
    return h_data;
};

template<typename T>
T* ImageData<T>::devicePtr(bool sync) {
    if(sync) {
        std::lock_guard<std::mutex> l_src(mtx);

        hipMemcpy(d_data, h_data, sizeof(T)*width*height, hipMemcpyHostToDevice);
    }
    return d_data;
};

template<typename T>
void ImageData<T>::copyTo(const ImageData<T>& dst) {
    std::lock_guard<std::mutex> l_src(mtx);
    std::lock_guard<std::mutex> l_dst(dst.mtx);
    hipMemcpy(dst.d_data, d_data, sizeof(T)*width*height, hipMemcpyDeviceToDevice);
};