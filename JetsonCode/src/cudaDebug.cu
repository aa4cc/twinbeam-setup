#include "cudaDebug.h"
#include "stdio.h"
#include "string.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

void gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
   if (code != hipSuccess) 
   {
      fprintf(stdout,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void cudaMemoryTest(const char *file, int line)
{
    const unsigned int N = 1048576;
    const unsigned int bytes = N * sizeof(int);
    int *h_a = (int*)malloc(bytes);
    int *d_a;
    gpuAssert(hipMalloc(&d_a, bytes), file, line);

    memset(h_a, 0, bytes);
    gpuAssert(hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice), file, line);
    gpuAssert(hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost), file, line);
}