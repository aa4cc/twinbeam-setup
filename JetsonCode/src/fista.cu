#include "hip/hip_runtime.h"
#include "fista.h"
#include "Phase_Kernels.h"
#include "Definitions.h"
#include <vector>
#include <iostream>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>
#include "stdio.h"
#include "cudaDebug.h"

Fista::Fista(
            double z,
            std::vector<double> rconstr,
            std::vector<double> iconstr,
            double mu,
            int width,
            int height,
            bool b_cost,
            double dx,
            double lambda,
            double n,
            hipStream_t stream
): width(width), height(height), b_cost(b_cost), mu(mu), rconstr(rconstr), iconstr(iconstr), z(z), stream(stream)
{

    count = width*height;

    allocate();
    propagator<<<N_BLOCKS, N_THREADS, 0, stream>>>(width, height, z, dx, n, lambda, Hq);
    conjugate<<<N_BLOCKS,N_THREADS, 0, stream>>>(count, Hq, Hn);
}

void Fista::allocate(){
    hipMalloc(&model, count*sizeof(hipfftDoubleComplex));
    hipMalloc(&Hq, count*sizeof(hipfftComplex));
    hipMalloc(&Hn, count*sizeof(hipfftComplex));
    hipMalloc(&propagation, count*sizeof(hipfftComplex));
    hipMalloc(&guess, count*sizeof(hipfftDoubleComplex));
    hipMalloc(&newGuess, count*sizeof(hipfftDoubleComplex));
    hipMalloc(&u, count*sizeof(hipfftDoubleComplex));
    hipMalloc(&temporary, count*sizeof(hipfftDoubleComplex));
    hipMalloc(&sumArr, 2*N_BLOCKS*sizeof(double));
    hipMalloc(&c, sizeof(double));
    hipMalloc(&image, count*sizeof(double));
    hipMalloc(&Imodel, count*sizeof(double));
    hipMalloc(&temporaryf, 2*count*sizeof(double));
    hipfftPlan2d(&fftPlan, height, width, HIPFFT_C2C);
    hipfftSetStream(fftPlan, stream);
}

void Fista::propagate(hipfftComplex* kernel, hipfftDoubleComplex* input, hipfftDoubleComplex* out){
    Z2C<<<N_BLOCKS,N_THREADS,0,stream>>>(count, input, propagation);
    hipfftExecC2C(fftPlan, propagation, propagation, HIPFFT_FORWARD);
    multiply<<<N_BLOCKS, N_THREADS,0,stream>>>(count, kernel, propagation);
    hipfftExecC2C(fftPlan, propagation, propagation, HIPFFT_BACKWARD);
    C2Z<<<N_BLOCKS,N_THREADS,0,stream>>>(count, propagation, out);
}

void Fista::calculateCost(double mu, double* model, hipfftDoubleComplex* guess, double* temp, double* out){
    absolute<<<N_BLOCKS,N_THREADS,0,stream>>>(count, guess, &temp[count]);
    square<<<N_BLOCKS,N_THREADS,0,stream>>>(count, model, &temp[count]);

    h_sum(count, &temp[count], sumArr, stream);
    h_sum(count, &temp[count], &sumArr[N_BLOCKS], stream);
    
    scalef<<<1,1,0,stream>>>(1,mu,sumArr,sumArr);
    simpleSum<<<1,1,0,stream>>>(&sumArr[N_BLOCKS],sumArr,&out[0]);
}

void Fista::normalize(int c, double* arr){
    h_minimum(c, arr, sumArr, stream);

    double temp;
    hipMemcpyAsync(&temp, sumArr, sizeof(double), hipMemcpyDeviceToHost, stream);
    offsetf<<<N_BLOCKS,N_THREADS,0,stream>>>(c, -temp, arr, arr, true);

    h_maximum(c, arr, sumArr, stream);
    contractf_p<<<N_BLOCKS,N_THREADS,0,stream>>>(c, sumArr, arr, arr);
}

void Fista::iterate(double *input, int iters, bool warm){
    // Initialization of variables
    s = 1;
    if(b_cost){
        hipMalloc(&cost, (1+iters)*sizeof(double));
        h_cost = (double*)malloc((iters+1)*sizeof(double));
    }

    //Copying the input image from host to device memory - computationally complex
    gpuErrchk(hipMemcpyAsync(image, input, count*sizeof(double), hipMemcpyHostToDevice, stream));
    //blur->gaussianBlur(width,height, 5, 3, image, temporaryf, image);
    h_average(count, image, sumArr, stream);
    contractf_p<<<N_BLOCKS,N_THREADS,0,stream>>>(count, sumArr, image, image);
    hipMemcpyAsync(m, sumArr, sizeof(double), hipMemcpyDeviceToHost, stream);

    //Copying the device memory image to device memory guesses

    if (!warm){
        F2C<<<N_BLOCKS,N_THREADS,0,stream>>>(count, image, u);
        F2C<<<N_BLOCKS,N_THREADS,0,stream>>>(count, image, guess);
    }
    
    for(int iter = 0; iter < iters; iter++){
        //Calculating the current iteration model 
        propagate(Hq, u, temporary);

        //Calculation of Imodel and model arrays
        modelFunc<<<N_BLOCKS,N_THREADS,0,stream>>>(count, 1.0f, 0, temporary, model, Imodel);

        //Calculation of the optimal scaling parameter c
        h_sumOfProducts(count, image, Imodel, sumArr, stream);
        h_sumOfProducts(count, Imodel, Imodel, &sumArr[N_BLOCKS], stream);
        contractf_p<<<1,1,0,stream>>>(1, &sumArr[N_BLOCKS], sumArr, c);
        double t_cost[1];
        hipMemcpyAsync(t_cost, c, sizeof(double), hipMemcpyDeviceToHost, stream);

        //Cost calculation with sparsity constraint
        linear<<<N_BLOCKS,N_THREADS,0,stream>>>(count, c, image, Imodel, temporaryf, false);

        if(b_cost){
            calculateCost(mu, temporaryf, guess, temporaryf, &cost[iter]);
            double t_cost[1];
            hipMemcpyAsync(t_cost, &cost[iter], sizeof(double), hipMemcpyDeviceToHost, stream);
            std::cout << "[DEBUG] Cost at iteration " << iter << " is " << t_cost[0] << std::endl;
        }

        //Calculating residues
        multiplyfc<<<N_BLOCKS,N_THREADS,0,stream>>>(count, temporaryf, model);
        propagate(Hn, model, temporary);

        double t = 0.2;
        scalef<<<1,1,0,stream>>>(1, 2*t, c, c);
        F2C<<<1,1,0,stream>>>(1,c,newGuess);
        scale_p<<<N_BLOCKS,N_THREADS,0,stream>>>(count, newGuess, temporary, temporary);
        add<<<N_BLOCKS,N_THREADS,0,stream>>>(count, u, temporary, newGuess, false);

        //Applying soft thresholding bounds
        softBounds<<<N_BLOCKS,N_THREADS,0,stream>>>(count, newGuess, mu, 1);

        //Applying strict bounds
        strictBounds<<<N_BLOCKS,N_THREADS,0,stream>>>(count, newGuess, rconstr[0], rconstr[1], iconstr[0], iconstr[1]);

        double s_new = 0.5*(1+std::sqrt(1+4*s*s));
        hipfftDoubleComplex temp = make_hipDoubleComplex((s-1)/s_new,0);
        add<<<N_BLOCKS,N_THREADS,0,stream>>>(count, newGuess, guess, temporary, false);
        scale<<<N_BLOCKS,N_THREADS,0,stream>>>(count, temp, temporary, temporary);
        add<<<N_BLOCKS,N_THREADS,0,stream>>>(count, newGuess, temporary, u, true);

        s = s_new;
        hipMemcpyAsync(guess, newGuess, count*sizeof(hipfftDoubleComplex), hipMemcpyDeviceToDevice, stream);
    
    }
    
    // Final cost calculation
    if(b_cost){
        propagate(Hq, u, newGuess);

        //Calculation of Imodel and model arrays
        modelFunc<<<N_BLOCKS,N_THREADS, 0, stream>>>(count, 1.0f, 0, newGuess, model, Imodel);

        //Calculation of the optimal scaling parameter c
        h_sumOfProducts(count, image, Imodel, sumArr, stream);
        h_sumOfProducts(count, Imodel, Imodel, &sumArr[N_BLOCKS], stream);
        contractf_p<<<1,1,0,stream>>>(1, &sumArr[N_BLOCKS], sumArr, c);

        //Cost calculation with sparsity constraint
        linear<<<N_BLOCKS,N_THREADS,0,stream>>>(count, c, image, Imodel, temporaryf, false);

        calculateCost(mu, temporaryf, guess, temporaryf, &cost[iters]);
        double t_cost[1];
        hipMemcpyAsync(t_cost, &cost[iters], sizeof(double), hipMemcpyDeviceToHost, stream);
        std::cout << "Current cost at iteration " << iters << " is " << t_cost[0] << std::endl;

        gpuErrchk(hipMemcpyAsync(h_cost, cost, (iters+1)*sizeof(double), hipMemcpyDeviceToHost, stream));
        hipFree(cost);
    }

    // Moving results to host memory
    // Adding one to get the light wavefront (otherwise we only have the disturbance by the particles and electrodes)
    offset<<<N_BLOCKS,N_THREADS, 0, stream>>>(count, 1.0f, 0.0f, guess, temporary);

    // Check if any error occured - important to note that untested kernels can lead to exceptions at hipMemcpy calls
    gpuErrchk(hipPeekAtLastError());
}

void Fista::update(uint8_t* modulus, uint8_t* phase){
    // temporary contains the latest results in complex form
    
    // Processing the modulus
    absolute<<<N_BLOCKS,N_THREADS, 0, stream>>>(count,temporary,temporaryf);
    scalef<<<N_BLOCKS,N_THREADS, 0, stream>>>(count, m[0], temporaryf, temporaryf);
    //normalize(count, temporaryf);
    D2u8<<<N_BLOCKS,N_THREADS, 0, stream>>>(count,temporaryf,modulus);

    // Processing the phase
    angle<<<N_BLOCKS,N_THREADS, 0, stream>>>(count,temporary,temporaryf);
    normalize(count, temporaryf);
    D2u8<<<N_BLOCKS,N_THREADS, 0, stream>>>(count,temporaryf,phase);
}

Fista::~Fista(){
    hipFree(Hq);
    hipFree(Hn);
    hipFree(temporary);
    hipFree(image);
    hipFree(model);
    hipFree(guess);
    hipFree(newGuess);
    hipFree(u);
    hipFree(temporaryf);
    hipFree(c);
    hipFree(propagation);
    hipfftDestroy(fftPlan);
}