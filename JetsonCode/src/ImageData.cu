/**
 * @author  Martin Gurtner
 */

#include "ImageData.h"

template<typename T>
bool ImageData<T>::create(uint16_t m, uint16_t n) {
    release();
    width = m; height = n;
    // Allocate memory on the host side
    if(hipHostAlloc((void **)&h_data,  sizeof(T)*width*height,  hipHostMallocMapped) != hipSuccess) return false;
    // Allocate memory on the device side
    if(hipMalloc((void **)&d_data,  sizeof(T)*width*height) != hipSuccess) return false;

    return true;
};

template<typename T>
void ImageData<T>::release() {
    if (h_data) hipHostFree(h_data);
    if (d_data) hipFree(d_data);
};

template<typename T>
T* ImageData<T>::hostPtr(bool sync) {
    if(sync) {
        std::shared_lock<std::shared_timed_mutex> lck(mtx);

        hipMemcpy(h_data, d_data, sizeof(T)*width*height, hipMemcpyDeviceToHost);
    }
    return h_data;
};

template<typename T>
T* ImageData<T>::hostPtrAsync(hipStream_t stream, bool sync) {
    if(sync) {
        std::shared_lock<std::shared_timed_mutex> lck(mtx);

        hipMemcpyAsync(h_data, d_data, sizeof(T)*width*height, hipMemcpyDeviceToHost, stream);
    }
    return h_data;
};

template<typename T>
T* ImageData<T>::devicePtr() {
    return d_data;
};

template<typename T>
void ImageData<T>::copyTo(const ImageData<T>& dst) {
    std::shared_lock<std::shared_timed_mutex> l_src(mtx);
    std::unique_lock<std::shared_timed_mutex> l_dst(dst.mtx);
    hipMemcpy(dst.d_data, d_data, sizeof(T)*width*height, hipMemcpyDeviceToDevice);
};

template<typename T>
void ImageData<T>::copyToAsync(const ImageData<T>& dst, hipStream_t stream) {
    std::shared_lock<std::shared_timed_mutex> l_src(mtx);
    std::unique_lock<std::shared_timed_mutex> l_dst(dst.mtx);
    hipMemcpyAsync(dst.d_data, d_data, sizeof(T)*width*height, hipMemcpyDeviceToDevice, stream);
};