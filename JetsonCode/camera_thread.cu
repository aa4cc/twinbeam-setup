#include "hip/hip_runtime.h"
/**
 * @author  Martin Gurtner
 * @author  Viktor-Adam Koropecky
 */
#include "camera_thread.h"
#include "Definitions.h"
#include "Kernels.h"
#include "hip/hip_runtime.h"
#include "cudaEGL.h"
#include "Argus/Argus.h"
#include "EGLStream/EGLStream.h"
#include "EGL/egl.h"
#include "cuda_egl_interop.h"
#include "argpars.h"
#include "CameraController.h"
#include <unistd.h>
#include <stdio.h>
#include <stdlib.h>

using namespace std;
using namespace Argus;
using namespace EGLStream;

hipError_t res;
hipArray_t yArray;
hipArray_t uvArray;
texture<unsigned char, 2, hipReadModeElementType> yTexRef;
texture<uchar2, 2, hipReadModeElementType> uvTexRef;
const textureReference* uvTex;
const textureReference* yTex;
hipChannelFormatDesc yChannelDesc;
hipChannelFormatDesc uvChannelDesc;

int numBlocks;

#define  CLAMP_F2UINT8(in) ((in) > 255 ? 255: (in))

// Converts the captured image in YUV format stored in yTexRef and uvTexRef to red and green channel stored in G and R arrays
// !Important: the y-axis is flipped and red channel is shifted with respect to the green channel by an offset.
__global__ void yuv2bgr(int width, int height, int offset_x, int offset_y,
						uint8_t* G, uint8_t* R)
        {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            int count = width*height;
            int tx, tx2, ty, ty2;
            float y1, y2;
            float u1, v1, v2;
            for (int i = index; i < count; i += stride)
            {
            	ty 	= height - i/width - 1 + offset_y;
            	ty2 = height - i/width - 1 + offset_y;
				tx 	= i%width + offset_x;
				tx2 = i%width + offset_x + (512);
            	y1 = (float)((tex2D<unsigned char>(yTexRef, (float)tx+0.5f, (float)ty+0.5f) - (float)16) * 1.164383f);
            	y2 = (float)((tex2D<unsigned char>(yTexRef, (float)tx2+0.5f, (float)ty2+0.5f) - (float)16) * 1.164383f);
            	u1 = (float)(tex2D<uchar2>(uvTexRef, (float)(tx/2)+(float)(tx%2)+0.5f,
					  (float)(ty/2)+(float)(ty%2)+0.5f).x - 128) * 0.391762f;
            	v2 = (float)(tex2D<uchar2>(uvTexRef, (float)(tx2/2)+(float)(tx2%2)+0.5f,
            	     (float)(ty2/2)+(float)(ty2%2)+0.5f).y - 128) * 1.596027f;
            	v1 = (float)(tex2D<uchar2>(uvTexRef, (float)(tx/2)+(float)(tx%2)+0.5f,
            	     (float)(ty/2)+(float)(ty%2)+0.5f).y - 128) * 0.812968f;
				G[i] = CLAMP_F2UINT8(y1-u1-v1);
				R[i] = CLAMP_F2UINT8(y2+v2);
            }
		}
		

// 		__global__ void yuv2bgr(int width, int height, int offset_x, int offset_y,
// 			uint8_t* G, uint8_t* R)
// {
// int index = blockIdx.x * blockDim.x + threadIdx.x;
// int stride = blockDim.x * gridDim.x;
// int count = width*height;
// int tx, ty, ty2;
// for (int i = index; i < count; i += stride)
// {
// 	ty = i/width + offset_y;
// 	ty2 = i/width + offset_y - (512);
// 	tx = i%width + offset_x;
	
// 	unsigned char Y_1  	= tex2D<unsigned char>(yTexRef, (float)tx+0.5f, (float)ty+0.5f);
// 	unsigned char Y_2  	= tex2D<unsigned char>(yTexRef, (float)tx+0.5f, (float)ty2+0.5f);
// 	uchar2 UV_1 		= tex2D<uchar2>(uvTexRef, (float)(tx/2)+(float)(tx%2)+0.5f, (float)(ty/2)+(float)(ty%2)+0.5f);
// 	uchar2 UV_2  		= tex2D<uchar2>(uvTexRef, (float)(tx/2)+(float)(tx%2)+0.5f,  (float)(ty2/2)+(float)(ty2%2)+0.5f);

// 	uint16_t C_1 =  (uint16_t)Y_1 		- 16;
// 	uint16_t D_1 =  (uint16_t)UV_1.x 	- 128;
// 	uint16_t E_1 =  (uint16_t)UV_1.y 	- 128;

// 	uint16_t C_2 =  (uint16_t)Y_2 		- 16;
// 	uint16_t E_2 =  (uint16_t)UV_2.y 	- 128;

// 	uint16_t G_u16 = (298*C_1 - 100*D_1 - 208*E_1 + 128) >> 8;
// 	uint16_t R_u16 = (298*C_2 + 409*E_2 + 128) >> 8;
// 	G[i] = CLAMP_U16_2_U8(G_u16);
// 	R[i] = CLAMP_U16_2_U8(R_u16);
// }
// }

void camera_thread(AppData& appData){
	printf("INFO: camera_thread: started\n");
	
	CameraController camController(0, 1, Options::verbose, Options::debug);
	if(!camController.Initialize()) {
		fprintf(stderr, "ERROR: Unable to initialize the camer!\n");
		appData.exitTheApp();
	}
	
	//CUDA variable declarations

	cudaEglStreamConnection conn;
	hipGraphicsResource_t resource;
	cudaEglFrame eglFrame;		
	hipArray_t yArray;
	hipArray_t uvArray;
	hipChannelFormatDesc yChannelDesc;
	hipChannelFormatDesc uvChannelDesc;
	
	while(!appData.appStateIs(AppData::AppState::EXITING)){
		if(Options::debug) printf("INFO: camera_thread: waiting for entering the INITIALIZING state\n");

		// Wait till the app enters the INITIALIZING state. If this fails (which could happen only in case of entering the EXITING state), break the loop.
		if(!appData.waitTillState(AppData::AppState::INITIALIZING)) break;

		// The app is in the INITIALIZING state
		// Initialize the camera
		if(!camController.Start(appData.values[STG_WIDTH], appData.values[STG_HEIGHT],appData.values[STG_FPS], appData.values[STG_EXPOSURE], appData.values[STG_ANALOGGAIN], appData.values[STG_DIGGAIN])) {
			fprintf(stderr, "ERROR: Unable to start capturing the images from the camera\n");
			appData.exitTheApp();
			break;				
		}

		res = cudaEGLStreamConsumerConnect(&conn, camController.GetEGLStream());
		if (res != hipSuccess) {
			fprintf(stderr, "ERROR: Unable to connect CUDA to EGLStream as a consumer\n");
			appData.exitTheApp();
			break;
		}

		appData.camIG.create(appData.values[STG_WIDTH], appData.values[STG_HEIGHT]);
		appData.camIR.create(appData.values[STG_WIDTH], appData.values[STG_HEIGHT]);
		
		numBlocks = 1024;
		
		yTexRef.normalized = 0;
		yTexRef.filterMode = hipFilterModePoint;
		yTexRef.addressMode[0] = hipAddressModeClamp;
		yTexRef.addressMode[1] = hipAddressModeClamp;
		hipGetTextureReference(&yTex, HIP_SYMBOL(&yTexRef));
		
		uvTexRef.normalized = 0;
		uvTexRef.filterMode = hipFilterModePoint;
		uvTexRef.addressMode[0] = hipAddressModeClamp;
		uvTexRef.addressMode[1] = hipAddressModeClamp;
		hipGetTextureReference(&uvTex, HIP_SYMBOL(&uvTexRef));
		
		// Set the flag indicating that the camera was initialized
		appData.camera_is_initialized = true;

		if(Options::debug) printf("INFO: camera_thread: waiting till other App components are initialized\n");

		// Wait till all the components of the App are initialized. If this fails, break the loop.
		if(!appData.waitTillAppIsInitialized()) break;

		// At this point, the app is in the AppData::AppState::RUNNING state.
		if(Options::debug) printf("INFO: camera_thread: entering the running stage\n");

		// Capture the images for as long as the App remains in the RUNNING state
		while(appData.appStateIs(AppData::AppState::RUNNING)){
			camController.NewFrameRequest();

			res = cudaEGLStreamConsumerAcquireFrame(&conn, &resource, 0, 5000);
			if(res != hipSuccess){
				continue;
			}
			cudaGraphicsResourceGetMappedEglFrame(&eglFrame, resource, 0, 0);
			yArray = eglFrame.frame.pArray[0];
			uvArray = eglFrame.frame.pArray[1];
			
			hipGetChannelDesc(&yChannelDesc, (hipArray_const_t)(yArray));
			hipBindTextureToArray(yTex, (hipArray_const_t)(yArray), &yChannelDesc);
			hipGetChannelDesc(&uvChannelDesc, (hipArray_const_t)(uvArray));
			hipBindTextureToArray(uvTex, (hipArray_const_t)(uvArray), &uvChannelDesc);

			numBlocks = (appData.get_area()/2 +NBLOCKS -1)/NBLOCKS;
			
			{
				std::lock_guard<std::mutex> lk(appData.cam_mtx);
				std::lock_guard<std::mutex> G_lk(appData.camIG.mtx);
				std::lock_guard<std::mutex> R_lk(appData.camIR.mtx);

				yuv2bgr<<<numBlocks, NBLOCKS>>>(appData.values[STG_WIDTH], appData.values[STG_HEIGHT],
												appData.values[STG_OFFSET_X], appData.values[STG_OFFSET_Y], appData.camIG.devicePtr(), appData.camIR.devicePtr());
			}
			appData.cam_cv.notify_all();

			hipUnbindTexture(yTex);
			hipUnbindTexture(uvTex);
			
			cudaEGLStreamConsumerReleaseFrame(&conn, resource, 0);
		}

		// Deinitialize the camera
		if(!camController.Stop()) {
			fprintf(stderr, "ERROR: Unable to stop capturing the images by the camera!\n");
			appData.exitTheApp();
		}					
		
		cudaEGLStreamConsumerDisconnect(&conn);

		// Set the flag indicating that the camera was initialized
		appData.camera_is_initialized = false;		
	}

	printf("INFO: camera_thread: ended\n");
}