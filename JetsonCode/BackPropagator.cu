/**
 * @author  Martin Gurtner
 * @author  Viktor-Adam Koropecky
 */
 
#include "BackPropagator.h"

BackPropagator::BackPropagator( int m, int n, float lambda, float backprop_dist ) :M{m}, N{n}
    {
        // Allocate memory for HQ and the image
        hipMalloc(&Hq, N*M*sizeof(hipfftComplex));
        hipMalloc(&image, N*M*sizeof(hipfftComplex));
        hipMalloc(&image_float, N*M*sizeof(float));

        // Declaring the FFT plan
        hipfftPlan2d(&fft_plan, N, M, HIPFFT_C2C);

        numBlocks = (m*n/2 + NBLOCKS -1)/NBLOCKS;

        // Calculating the Hq matrix according to the equations in the original .m file.
        calculateBackPropMatrix<<<numBlocks, NBLOCKS>>>(N, M, backprop_dist, PIXEL_DX, REFRACTION_INDEX, lambda, Hq);
    };

void BackPropagator::backprop(ImageData<uint8_t>& input, ImageData<uint8_t>& output)
{
    // Convert the uint8 image to float image 
    {
        std::lock_guard<std::mutex> l_src(input.mtx);
        u8ToFloat<<<numBlocks, NBLOCKS>>>(M, N, input.devicePtr(), image_float);
    }

    // Convert the real input image to complex image
    convertToComplex<<<numBlocks, NBLOCKS>>>(N*M, image_float, image);
    
    // Execute forward FFT on the green channel
    hipfftExecC2C(fft_plan, image, image, HIPFFT_FORWARD);

    // Element-wise multiplication of Hq matrix and the image
	multiplyInPlace<<<numBlocks, NBLOCKS>>>(M, N, Hq, image);
    
	// Executing inverse FFT
	hipfftExecC2C(fft_plan, image, image, HIPFFT_BACKWARD);
	// Conversion of result matrix to a real float matrix
	absoluteValue<<<numBlocks, NBLOCKS>>>(M,N, image, image_float);
    // Conversion of result matrix to a real float matrix
    std::lock_guard<std::mutex> l_src(output.mtx);
    floatToUInt8<<<numBlocks, NBLOCKS>>>(M,N, image_float, output.devicePtr());
}

BackPropagator::~BackPropagator() {
    hipFree(Hq);
    hipFree(image);
    hipFree(image_float);
    hipfftDestroy(fft_plan);
};