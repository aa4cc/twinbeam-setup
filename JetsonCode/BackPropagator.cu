#include "BackPropagator.h"

BackPropagator::BackPropagator( int m, int n, float lambda, float backprop_dist ) :M{m}, N{n}
    {
        // Allocate memory for HQ and the image
        hipMalloc(&Hq, N*M*sizeof(hipfftComplex));
        hipMalloc(&image, N*M*sizeof(hipfftComplex));

        // Declaring the FFT plan
        hipfftPlan2d(&fft_plan, N, M, HIPFFT_C2C);

        // Calculating the Hq matrix according to the equations in the original .m file.
        calculate<<<numBlocks, BLOCKSIZE>>>(N, M, backprop_dist, PIXEL_DX, REFRACTION_INDEX, lambda, Hq);
    };

void BackPropagator::backprop(float* input, float* output )
{
    // Convert the real input image to complex image
    convertToComplex<<<numBlocks, BLOCKSIZE>>>(N*M, input, image);
    
    // Execute forward FFT on the green channel
    hipfftExecC2C(fft_plan, image, image, HIPFFT_FORWARD);

    // Element-wise multiplication of Hq matrix and the image
	multiplyInPlace<<<numBlocks, BLOCKSIZE>>>(M, N, Hq, image);
    
	// Executing inverse FFT
	hipfftExecC2C(fft_plan, image, image, HIPFFT_BACKWARD);
	// Conversion of result matrix to a real double matrix
	imaginary<<<numBlocks, BLOCKSIZE>>>(M,N, image, output);
}

BackPropagator::~BackPropagator() {
    hipFree(Hq);
    hipFree(image);
    hipfftDestroy(fft_plan);
};