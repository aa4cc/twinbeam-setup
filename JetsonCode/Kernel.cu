#include "Kernel.h"
#include "Settings.h"
#include "Kernels.h"
#include "hipfft/hipfft.h"
#include "hip/hip_runtime.h"
#include "Definitions.h"

int numBlocks = 

void Kernel::allocate(){
	hipMalloc(&kernel, Settings::get_area()*sizeof(hipfftComplex));
}

void Kernel::deallocate(){
	hipFree(kernel);
}

void Kernel::set(hipfftComplex* new_kernel){
	kernel = new_kernel;
}

void Kernel::setInPhase(int dim, float* new_kernel){
	kernelToImage<<<numBlocks, BLOCKSIZE>>>(STG_WIDTH, STG_HEIGHT, dim, new_kernel, kernel);
	hipfftHandle plan;
    hipfftPlan2d(&plan, STG_HEIGHT, STG_WIDTH, HIPFFT_C2C);
    hipfftExecC2C(plan, kernel, kernel, HIPFFT_FORWARD);
    hipfftDestroy(plan);
}

void Kernel::update(hipfftComplex* new_kernel){
	multiplyInPlace<<<numBlocks, BLOCKSIZE>>>(STG_WIDTH, STG_HEIGHT, new_kernel, kernel);
}

void Kernel::updateInPhase(int dim, float* new_kernel){
	hipfftComplex* temporaryKernel;
	hipMalloc(&temporaryKernel, Settings::get_area()*sizeof(hipfftComplex));
	kernelToImage<<<numBlocks, BLOCKSIZE>>>(STG_WIDTH, STG_HEIGHT, dim, new_kernel, temporaryKernel);
	hipfftHandle plan;
    hipfftPlan2d(&plan, STG_HEIGHT, STG_WIDTH, HIPFFT_C2C);
    hipfftExecC2C(plan, temporaryKernel, temporaryKernel, HIPFFT_FORWARD);
    hipfftDestroy(plan);
    multiplyInPlace<<<numBlocks, BLOCKSIZE>>>(STG_WIDTH, STG_HEIGHT, temporaryKernel, kernel);
    hipFree(temporaryKernel);
}