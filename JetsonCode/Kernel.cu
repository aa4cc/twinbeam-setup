#include <cstdint>
#include "Kernel.h"
#include "Settings.h"
#include "Kernels.h"
#include "hipfft/hipfft.h"
#include "hip/hip_runtime.h"
#include "Definitions.h"

int numBlocksK = (Settings::get_area()/2 +BLOCKSIZE -1)/BLOCKSIZE; 

void Kernel::allocate(){
	hipMalloc(&kernel, Settings::get_area()*sizeof(hipfftComplex));
}

void Kernel::deallocate(){
	hipFree(kernel);
}

void Kernel::set(hipfftComplex* new_kernel){
	kernel = new_kernel;
}

void Kernel::setInPhase(int dim, float* new_kernel){
	kernelToImage<<<numBlocksK, BLOCKSIZE>>>(STG_WIDTH, STG_HEIGHT, dim, new_kernel, kernel);
	hipfftHandle plan;
    hipfftPlan2d(&plan, STG_HEIGHT, STG_WIDTH, HIPFFT_C2C);
    hipfftExecC2C(plan, kernel, kernel, HIPFFT_FORWARD);
    hipfftDestroy(plan);
}

void Kernel::update(hipfftComplex* new_kernel){
	multiplyInPlace<<<numBlocksK, BLOCKSIZE>>>(STG_WIDTH, STG_HEIGHT, new_kernel, kernel);
}

void Kernel::updateInPhase(int dim, float* new_kernel){
	hipfftComplex* temporaryKernel;
	hipMalloc(&temporaryKernel, Settings::get_area()*sizeof(hipfftComplex));
	kernelToImage<<<numBlocksK, BLOCKSIZE>>>(STG_WIDTH, STG_HEIGHT, dim, new_kernel, temporaryKernel);
	hipfftHandle plan;
    hipfftPlan2d(&plan, STG_HEIGHT, STG_WIDTH, HIPFFT_C2C);
    hipfftExecC2C(plan, temporaryKernel, temporaryKernel, HIPFFT_FORWARD);
    hipfftDestroy(plan);
    multiplyInPlace<<<numBlocksK, BLOCKSIZE>>>(STG_WIDTH, STG_HEIGHT, temporaryKernel, kernel);
    hipFree(temporaryKernel);
}